#include "inv.h"
void basedtinv(float* t,const int m,const int n,const int tupe,float* invA){
	int bat = m*n;
	hipfftComplex* t_f = (hipfftComplex*)malloc(bat*tupe*sizeof(hipfftComplex));
	//transform
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
	hipMemcpy(d_fftData,t_f,bat*tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	hipfftHandle plan;
	if(hipfftPlan1d(&plan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(plan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}

	//transform
	hipMemcpy(t_f,d_fftData,sizeof(hipfftComplex)*bat*tupe,hipMemcpyDeviceToHost);
	hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
	
/*printf("\n============================\n");
for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f2[i].x,t_f2[i].y);
}	
printf("\n============================\n");
*/	
    
    if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
		
	if(t_f != NULL){
	free(t_f);
	t_f = NULL;
	}
/*	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
*/
    //getrf
    int* Pivot;
    int* info;
    int* info_h = (int *)malloc(tupe*sizeof(int));
    hipComplex** Aarray_d;
    hipComplex** Ainv_d;
    hipComplex** Ainv_h;
    hipMalloc((void**)&Aarray_d,sizeof(hipComplex*));
    hipComplex** Aarray_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Aarray_h[0],sizeof(hipComplex)*bat);
    
    hipMalloc((void**)&Pivot,tupe*n*sizeof(int));
    hipMalloc((void**)&info,tupe*sizeof(int));

    hipMalloc((void**)&Ainv_d,sizeof(hipComplex*));
    Ainv_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Ainv_h[0],bat*sizeof(hipComplex));
    hipMemcpy(Ainv_d,Ainv_h,sizeof(hipComplex*),hipMemcpyHostToDevice);
    
	hipComplex* h_fftData = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);
    
    hipblasHandle_t handle;
    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasCreate error!",__FUNCTION__,__LINE__);
		return;
    }
    for(int i=0;i<tupe;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle,n,Aarray_d,n,Pivot+i*n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle,n,Aarray_d,n,Pivot+i*n,Ainv_d,n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

    }
    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasDestroy error!",__FUNCTION__,__LINE__);
		return;
    }
    //transform
	hipComplex* h_fftData1 = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		h_fftData1[i*tupe+j]=h_fftData[j*bat+i];
		}
	} 
	hipMemcpy(d_fftData, h_fftData1, sizeof(hipComplex)*tupe*bat,hipMemcpyHostToDevice);
    //delete ptr
   if(Aarray_h !=NULL){ 
        hipFree(Aarray_h[0]);
        Aarray_h=NULL;
        free(Aarray_h);
        Aarray_h=NULL;
    }
    if(Aarray_d !=NULL){ 
        hipFree(Aarray_d);
        Aarray_d=NULL;
    }
   if(Ainv_h != NULL){
        hipFree(Ainv_h[0]);
        Ainv_h=NULL;
        free(Ainv_h);
        Ainv_h=NULL;
    }	
    if(Ainv_d != NULL){
        hipFree(Ainv_d);
        Ainv_d=NULL;
    }
    if(Pivot !=NULL){
        hipFree(Pivot);
        Pivot=NULL;
    }
    if(info != NULL){
        hipFree(info);
        info=NULL;
    }
	if(h_fftData != NULL){
	free(h_fftData);
	h_fftData = NULL;
	}

	if(h_fftData1 != NULL){
	free(h_fftData1);
	h_fftData1 = NULL;
	}

	hipfftHandle iplan;

	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(iplan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
    }
    
	hipDeviceSynchronize();

	hipComplex* h_A = (hipComplex*)malloc(sizeof(hipComplex)*tupe*bat);
	hipMemcpy(h_A,d_fftData,tupe*bat*sizeof(hipComplex),hipMemcpyDeviceToHost);
	
	//transform
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		invA[j*bat+i]=h_A[i*tupe+j].x/tupe;
		}
	}
    if(d_fftData != NULL){
        hipFree(d_fftData);
        d_fftData =NULL;
    }
    if(h_A != NULL){
        free(h_A);
        h_A=NULL;
    }
	
}
