#include "based.h"
#include "inv.h"
void streamedtinv(float* t,const int m,const int n,const int tupe,float* invA){
#if 1
    //tfft:R2C
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	//estimate of the work size
	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}

    hipComplex* t_f2 = (hipComplex*)malloc(sizeof(hipComplex)*bat*ht);
    hipMemcpy(t_f2,d_fftData,sizeof(hipComplex)*bat*ht,hipMemcpyDeviceToHost);
    
    if(d_fftData != NULL){
	hipFree(d_fftData);
	d_fftData=NULL;	
    }
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }
    //getrf
    int* Pivot;
    int* info;
    int* info_h = (int *)malloc(ht*sizeof(int));
    hipComplex** Aarray_d;
    hipComplex** Ainv_d;
    hipComplex** Ainv_h;
    hipMalloc((void**)&Aarray_d,sizeof(hipComplex*));
    hipComplex** Aarray_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Aarray_h[0],sizeof(hipComplex)*bat);
    
    hipMalloc((void**)&Pivot,ht*n*sizeof(int));
    hipMalloc((void**)&info,ht*sizeof(int));

    hipMalloc((void**)&Ainv_d,sizeof(hipComplex*));
    Ainv_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Ainv_h[0],bat*sizeof(hipComplex));
    hipMemcpy(Ainv_d,Ainv_h,sizeof(hipComplex*),hipMemcpyHostToDevice);
    
    hipComplex* h_fftData = (hipComplex*)malloc(sizeof(hipComplex) * bat * ht);
    
    hipblasHandle_t* handle = (hipblasHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipblasHandle_t));
    memset(handle,0,sizeof(hipblasHandle_t));
    for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipblasCreate(&handle[i]) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasCreate error!",__FUNCTION__,__LINE__);
		return;
    }
       if( hipblasSetStream(handle[i],stream[i]) != HIPBLAS_STATUS_SUCCESS){
            fprintf(stdout,"[%s]:[%d] cubalsSetStream error!",__FUNCTION__,__LINE__);
            return;
       }
    }
    int tupe_num = ht/PLAN1D_SIZE;
    int tupe_s = ht%PLAN1D_SIZE;
    if( tupe_num > 0){
    for(int j=0;j < tupe_num; j++){
    for(int i=0;i<PLAN1D_SIZE;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat+j*bat*PLAN1D_SIZE,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n+j*n*PLAN1D_SIZE,info+i+j*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n+j*n*PLAN1D_SIZE,Ainv_d,n,info+i+j*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat+j*bat*PLAN1D_SIZE,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);
        }
        }
    for(int i=0;i<tupe_s;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat+tupe_num*bat*PLAN1D_SIZE,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n+tupe_num*n*PLAN1D_SIZE,info+i+tupe_num*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n+tupe_num*n*PLAN1D_SIZE,Ainv_d,n,info+i+tupe_num*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat+tupe_num*bat*PLAN1D_SIZE,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);

        }
    }else{
    for(int i=0;i<tupe_s;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n,Ainv_d,n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);

        }
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipblasDestroy(handle[i]) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasDestroy error!",__FUNCTION__,__LINE__);
		return;
        }
    }

    hipComplex* d_ifftData;
    hipMalloc((void**)&d_ifftData,sizeof(hipComplex)*bat*ht);
    hipMemcpy(d_ifftData,h_fftData,sizeof(hipComplex)*bat*ht,hipMemcpyHostToDevice);

    //delete ptr
   if(Aarray_h !=NULL){ 
        hipFree(Aarray_h[0]);
        Aarray_h=NULL;
        free(Aarray_h);
        Aarray_h=NULL;
    }
    if(Aarray_d !=NULL){ 
        hipFree(Aarray_d);
        Aarray_d=NULL;
    }
   if(Ainv_h != NULL){
        hipFree(Ainv_h[0]);
        Ainv_h=NULL;
        free(Ainv_h);
        Ainv_h=NULL;
    }	
    if(Ainv_d != NULL){
        hipFree(Ainv_d);
        Ainv_d=NULL;
    }
    if(Pivot !=NULL){
        hipFree(Pivot);
        Pivot=NULL;
    }
    if(info != NULL){
        hipFree(info);
        info=NULL;
    }
	if(h_fftData != NULL){
	free(h_fftData);
	h_fftData = NULL;
	}
//ifft
	hipfftHandle iplan =0;
	in[0] = ht;
	on[0] = tupe;
	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));
	
	float* d_inv;
	hipMalloc((void**)&d_inv,sizeof(float)*tupe*bat);

	if(hipfftExecC2R(iplan,(hipfftComplex*)d_ifftData,d_inv)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
       	int num=bat*tupe;
	float* invA_temp = (float*)malloc(sizeof(float)*tupe*bat);

	hipMemcpy(invA_temp,d_inv,sizeof(float)*bat*tupe,hipMemcpyDeviceToHost);
	for(int i=0;i<num;i++){
		invA[i]=invA_temp[i]/tupe;
	}
	hipFree(d_ifftData);        
	hipFree(d_inv);
#endif	
}
