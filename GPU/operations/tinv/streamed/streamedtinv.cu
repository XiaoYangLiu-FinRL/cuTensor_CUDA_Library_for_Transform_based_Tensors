#include "based.h"
#include "inv.h"
void streamedtinv(float* t,const int m,const int n,const int tupe,float* invA){
    int bat =m*n;
    hipfftComplex* t_f;

    hipHostAlloc((void**)&t_f,bat*tupe*sizeof(hipfftComplex),hipHostMallocDefault);
    
    //transform t1
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
    //process bat
    int bat_num = bat/PLAN1D_SIZE;
    int bat_s = bat%PLAN1D_SIZE;
	hipfftHandle * plan=(hipfftHandle*)malloc(sizeof(hipfftHandle)*PLAN1D_SIZE);
    memset(plan,0,sizeof(hipfftHandle));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&plan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipfftSetStream(plan[i],stream[i]);
    }
    if(bat_num > 0){
    for(int j=0;j<bat_num;j++){

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,t_f+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
        return;
    }
     }

    #pragma unroll	
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
            	}
        	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(t_f+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
        return;
    }
         }
    }

    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+PLAN1D_SIZE*bat_num*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+tupe*bat_num*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }else{
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe,t_f+i*tupe,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe,d_fftData+i*tupe,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
	//transform
    hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
//    printf("\n============================\n");
//    for(int i=0;i<bat*tupe;i++){
//    printf("[%f %f]",t_f2[i].x,t_f2[i].y);
//    }	
//    printf("\n============================\n");
    	
    
    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipfftDestroy(plan[i])!=HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }
    }
		
	if(t_f != NULL){
	hipHostFree(t_f);
	t_f = NULL;
	}
    
    //getrf
    int* Pivot;
    int* info;
    int* info_h = (int *)malloc(tupe*sizeof(int));
    hipComplex** Aarray_d;
    hipComplex** Ainv_d;
    hipComplex** Ainv_h;
    hipMalloc((void**)&Aarray_d,sizeof(hipComplex*));
    hipComplex** Aarray_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Aarray_h[0],sizeof(hipComplex)*bat);
    
    hipMalloc((void**)&Pivot,tupe*n*sizeof(int));
    hipMalloc((void**)&info,tupe*sizeof(int));

    hipMalloc((void**)&Ainv_d,sizeof(hipComplex*));
    Ainv_h=(hipComplex**)malloc(sizeof(hipComplex*));
    hipMalloc((void**)&Ainv_h[0],bat*sizeof(hipComplex));
    hipMemcpy(Ainv_d,Ainv_h,sizeof(hipComplex*),hipMemcpyHostToDevice);
    
	hipComplex* h_fftData = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);
    
    hipblasHandle_t* handle = (hipblasHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipblasHandle_t));
    memset(handle,0,sizeof(hipblasHandle_t));
    for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipblasCreate(&handle[i]) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasCreate error!",__FUNCTION__,__LINE__);
		return;
    }
       if( hipblasSetStream(handle[i],stream[i]) != HIPBLAS_STATUS_SUCCESS){
            fprintf(stdout,"[%s]:[%d] cubalsSetStream error!",__FUNCTION__,__LINE__);
            return;
       }
    }
    int tupe_num = tupe/PLAN1D_SIZE;
    int tupe_s = tupe%PLAN1D_SIZE;
    if( tupe_num > 0){
    for(int j=0;j < tupe_num; j++){
    for(int i=0;i<PLAN1D_SIZE;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat+j*bat*PLAN1D_SIZE,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n+j*n*PLAN1D_SIZE,info+i+j*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n+j*n*PLAN1D_SIZE,Ainv_d,n,info+i+j*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat+j*bat*PLAN1D_SIZE,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);
        }
        }
    for(int i=0;i<tupe_s;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat+tupe_num*bat*PLAN1D_SIZE,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n+tupe_num*n*PLAN1D_SIZE,info+i+tupe_num*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n+tupe_num*n*PLAN1D_SIZE,Ainv_d,n,info+i+tupe_num*PLAN1D_SIZE,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat+tupe_num*bat*PLAN1D_SIZE,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);

        }
    }else{
    for(int i=0;i<tupe_s;i++){
        if(hipMemcpy(Aarray_h[0],t_f2+i*bat,sizeof(hipfftComplex)*bat,hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipMemcpy(Aarray_d,Aarray_h,sizeof(hipComplex*),hipMemcpyHostToDevice) != hipSuccess){
		    fprintf(stdout,"[%s]:[%d] cudeMemcpy failed!",__FUNCTION__,__LINE__);
		    return;
        }
        if(hipblasCgetrfBatched(handle[i],n,Aarray_d,n,Pivot+i*n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetrf error!",__FUNCTION__,__LINE__);
		return;
        }

        hipMemcpy(info_h,info,sizeof(int),hipMemcpyDeviceToHost);
        printf("[ %d ] ",info_h[0]);
        hipDeviceSynchronize();
        if(hipblasCgetriBatched(handle[i],n,Aarray_d,n,Pivot+i*n,Ainv_d,n,info+i,1) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cublasCgetri error!",__FUNCTION__,__LINE__);
		return;
        }
        hipMemcpy(h_fftData+i*bat,Ainv_h[0],bat*sizeof(hipComplex),hipMemcpyDeviceToHost);

        }
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipblasDestroy(handle[i]) != HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipblasDestroy error!",__FUNCTION__,__LINE__);
		return;
        }
    }
    //transform
	hipComplex* h_fftData1;
    hipHostAlloc((void**)&h_fftData1,bat*tupe*sizeof(hipComplex),hipHostMallocDefault);
    
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		h_fftData1[i*tupe+j]=h_fftData[j*bat+i];
		}
	} 
    //delete ptr
   if(Aarray_h !=NULL){ 
        hipFree(Aarray_h[0]);
        Aarray_h=NULL;
        free(Aarray_h);
        Aarray_h=NULL;
    }
    if(Aarray_d !=NULL){ 
        hipFree(Aarray_d);
        Aarray_d=NULL;
    }
   if(Ainv_h != NULL){
        hipFree(Ainv_h[0]);
        Ainv_h=NULL;
        free(Ainv_h);
        Ainv_h=NULL;
    }	
    if(Ainv_d != NULL){
        hipFree(Ainv_d);
        Ainv_d=NULL;
    }
    if(Pivot !=NULL){
        hipFree(Pivot);
        Pivot=NULL;
    }
    if(info != NULL){
        hipFree(info);
        info=NULL;
    }
	if(h_fftData != NULL){
	free(h_fftData);
	h_fftData = NULL;
	}

	hipComplex* h_A;
    hipHostAlloc((void**)&h_A,bat*tupe*sizeof(hipComplex),hipHostMallocDefault);
	
    hipfftHandle* iplan=(hipfftHandle*)malloc(PLAN1D_SIZE*sizeof(hipfftHandle));
    memset(iplan,0,sizeof(hipfftHandle));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&iplan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
        hipfftSetStream(iplan[i],stream[i]);
    }
    bat_num = bat/PLAN1D_SIZE;
    bat_s = bat%PLAN1D_SIZE;
    if(bat_num > 0){ 
	for(int j=0;j<bat_num;j++){
	for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE, h_fftData1+i*tupe+j*tupe*PLAN1D_SIZE, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(h_A+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
	}
    }
	for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE, h_fftData1+i*tupe+bat_num*tupe*PLAN1D_SIZE, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(h_A+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
	}
    }else{
	for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe, h_fftData1+i*tupe, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(h_A+i*tupe,d_fftData+i*tupe,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
	}
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
	//transform
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		invA[j*bat+i]=h_A[i*tupe+j].x/tupe;
		}
	}
	for(int i=0;i<PLAN1D_SIZE;i++){
	    if(hipfftDestroy(iplan[i])!=HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }
	    if(hipStreamDestroy(stream[i])!= hipSuccess){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }

    }
    if(h_fftData1 != NULL){
	hipHostFree(h_fftData1);
	h_fftData1 = NULL;
	}
    if(d_fftData != NULL){
        hipFree(d_fftData);
        d_fftData =NULL;
    }
    if(h_A != NULL){
        hipHostFree(h_A);
        h_A=NULL;
    }
	
}
