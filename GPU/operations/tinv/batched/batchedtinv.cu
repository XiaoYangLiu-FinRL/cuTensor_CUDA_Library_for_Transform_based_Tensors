#include"inv.h" 
void batchedtinv(float* t,const int m,const int n,const int tupe,float* invA){
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	//estimate of the work size
	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
	
	
	magma_init();
	magma_queue_t queue = NULL;
	magma_int_t dev = 0;
	magma_queue_create( dev, &queue);


	magmaFloatComplex *d_A, *d_invA;
	magmaFloatComplex_ptr *dA_array;
	magmaFloatComplex_ptr *dinvA_array;
	magma_int_t **dipiv_array;
	magma_int_t *dinfo_array;
	magma_int_t *d_ipiv, *d_info;
	magma_int_t M, N, lda, ldda;
	magma_int_t columns;

	M = m;
	N = n;
	magma_int_t batchCount = ht;
	lda = M;
	ldda = magma_roundup( M, 32 );

	magma_cmalloc( &d_A,      ldda*N * batchCount );
	magma_cmalloc( &d_invA,   ldda*N * batchCount );
	magma_imalloc( &d_ipiv,   N * batchCount );
	magma_imalloc( &d_info,   batchCount );

	magma_malloc( (void**) &dA_array,   batchCount * sizeof(magmaFloatComplex*));
	magma_malloc( (void**) &dinvA_array,batchCount * sizeof(magmaFloatComplex*));
	magma_malloc( (void**) &dipiv_array,batchCount * sizeof(magma_int_t*));
	magma_imalloc( &dinfo_array, batchCount);
	
	columns = N * batchCount;
	
	magma_ccopymatrix( M, columns, d_fftData, lda, d_A, ldda, queue );
	
	magma_cset_pointer( dA_array, d_A, ldda, 0, 0, ldda * N, batchCount, queue );
	magma_cset_pointer( dinvA_array, d_invA, ldda, 0, 0, ldda * N, batchCount, queue );
	magma_iset_pointer( dipiv_array, d_ipiv, 1, 0, 0, N, batchCount, queue );

//	magma_cprint_gpu( M, columns, d_A, ldda, queue );

	
	magma_cgetrf_batched( M, N, dA_array, ldda, dipiv_array, dinfo_array, batchCount, queue );
	magma_cgetri_outofplace_batched( M, dA_array, ldda, dipiv_array, dinvA_array, ldda, dinfo_array, batchCount, queue );
	
//	magma_cprint_gpu( M, columns, d_invA, ldda, queue );
	
	magma_ccopymatrix( M, columns, d_invA, ldda, d_fftData, lda, queue );
	
	magma_queue_destroy( queue );
	magma_free( d_A   );
        magma_free( d_invA );
        magma_free( dA_array   );
        magma_free( dinvA_array  );
	magma_free( d_ipiv );
	magma_free( d_info );
	magma_free( dipiv_array );
	magma_free( dinfo_array );
        magma_finalize();
	
	hipfftHandle iplan =0;
	in[0] = ht;
	on[0] = tupe;
	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));
	
	float* d_inv;
	hipMalloc((void**)&d_inv,sizeof(float)*tupe*bat);
	if(hipfftExecC2R(iplan,(hipfftComplex*)d_fftData,d_inv)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

       	int num=bat*tupe;
	int threads,blocks;
        if(num<512){
          threads=num;
          blocks=1;
        }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads>>>(d_inv,num,tupe);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(invA,d_inv,sizeof(float)*bat*tupe,hipMemcpyDeviceToHost);
        
	hipFree(d_inv);
	hipFree(d_fftData); 
        
	}
