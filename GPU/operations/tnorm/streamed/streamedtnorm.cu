#include "hip/hip_runtime.h"
#include "norm.h"
#include "based.h"
/**
* normlization of tnesor t:
*       t = v * a;
* ||v|| = 1,it's mean <v,v> = e.
* INPUT: t is m×1×n
*
* OUTPUT: v*a  = t.
*/
void streamedtnorm(float* t, const int m, const int n, float* v, float* a){
	int ht = n/2+1;
	int bat = m;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*n);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*m*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*n,hipMemcpyHostToDevice);	
	
	//tfft
	hipfftHandle plan;
	int n_f[1] = {n};
	int in[1] = {n};
	int ou[1] = {ht}; 
	int stride_in = bat,dist_in = 1;
	int stride_ou = bat,dist_ou = 1;
	
	if(hipfftPlanMany(&plan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_R2C,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	} 
	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
	//solve normlize
	int threads =512;
	int blocks = 1;	
	//set stream
	
	hipStream_t* stream = (hipStream_t*)malloc(sizeof(hipStream_t)*PLAN1D_SIZE);	

	#pragma unroll
	for(int i=0;i<PLAN1D_SIZE;i++){
	   hipStreamCreate(&stream[i]);
	}

	hipComplex *d_hv,*d_ha;
	hipMalloc((void**)&d_hv,sizeof(hipComplex)*m*ht);
	hipMalloc((void**)&d_ha,sizeof(hipComplex)*ht);
	
	int tube_num = ht/PLAN1D_SIZE;
	int tube_s = ht%PLAN1D_SIZE;
	if(tube_num > 0){
	 for(int j=0;j< tube_num;j++){
	   for(int i=0;i<PLAN1D_SIZE;i++){
	      d_normlize<<<blocks,threads,0,stream[i]>>>(d_fftData+i*m+j*m*PLAN1D_SIZE,m,d_hv+i*m+j*m*PLAN1D_SIZE,d_ha+i+j*PLAN1D_SIZE);
	   }
	   }
	
	for(int i=0;i<tube_s;i++){
	     d_normlize<<<blocks,threads,0,stream[i]>>>(d_fftData+i*m+tube_num*m*PLAN1D_SIZE,m,d_hv+i*m+tube_num*m*PLAN1D_SIZE,d_ha+i+tube_num*PLAN1D_SIZE);
	}
	}else{
	for(int i=0;i<tube_s;i++){
	     d_normlize<<<blocks,threads,0,stream[i]>>>(d_fftData+i*m,m,d_hv+i*m,d_ha+i);
	}
	}

	#pragma unroll
	for(int i=0;i<PLAN1D_SIZE;i++){
	   hipStreamSynchronize(stream[i]);
	}

	#pragma unroll
	for(int i=0;i<PLAN1D_SIZE;i++){
	   hipStreamDestroy(stream[i]);
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipFree(d_fftData);	
	//d_hv and d_ha take ifft 
//	int threads = 0;
//	int blocks = 0;	
	int num = 0;
	
	float *d_v,*d_a;
	hipMalloc((void**)&d_v,sizeof(float)*m*n);
	hipMalloc((void**)&d_a,sizeof(float)*n);
 
	hipfftHandle iplan;
	in[0] = ht;
	ou[0] = n;

	if(hipfftPlanMany(&iplan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_C2R,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	} 
	if(hipfftExecC2R(iplan,d_hv,d_v)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	num = m*n;
	if(num < 512){
	    threads = num;
	    blocks = 1;
	}else{
	    threads = 512;
	    blocks = ((num%512) == 0)?num/512:num/512+1;
	}

	fftResultProcess<<<blocks,threads>>>(d_v,num,n);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(v,d_v,sizeof(float)*m*n,hipMemcpyDeviceToHost);
	
	hipFree(d_hv);
	hipFree(d_v);
	
	stride_in = 1;
	stride_ou = 1;
	dist_in = 1;
	dist_ou = 1;
	bat = 1;

	if(hipfftPlanMany(&iplan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_C2R,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftExecC2R(iplan,d_ha,d_a)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
 	}	
	num = n;
	if(n < 512){
	    threads = num;
	    blocks = 1;
	}else{
	    threads = 512;
	    blocks = ((num%512) == 0)?num/512:num/512+1;
	}	

	fftResultProcess<<<blocks,threads>>>(d_a,num,n);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(a,d_a,sizeof(float)*n,hipMemcpyDeviceToHost);
	
	hipFree(d_ha);	
	hipFree(d_a);
}
