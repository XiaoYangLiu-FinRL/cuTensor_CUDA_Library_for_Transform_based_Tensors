#include "hip/hip_runtime.h"
#include "norm.h"
#include "based.h"
void basedtnorm(float* t,const int m,const int n,float* v,float* a){
	int bat = m;
	int tube = n;
	hipfftComplex* t_f = (hipfftComplex*)malloc(sizeof(hipfftComplex)*m*n);
	//transform forward
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tube;j++){
		t_f[i*tube+j].x = t[j*bat+i];
		t_f[i*tube+j].y = 0;
	  }
	}

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*m*n);
	hipMemcpy(d_fftData,t_f,sizeof(hipComplex)*m*n,hipMemcpyHostToDevice);
	
	hipfftHandle plan;
	if(hipfftPlan1d(&plan,tube,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(plan,d_fftData+i*tube,d_fftData+i*tube,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	
	//transform inverse

	hipMemcpy(t_f,d_fftData,sizeof(hipComplex)*m*n,hipMemcpyDeviceToHost);
	hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipComplex)*m*n);
	
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tube;j++){
		t_f2[j*bat+i]=t_f[i*tube+j];
	  }
	}
	
	hipMemcpy(d_fftData,t_f2,sizeof(hipComplex)*m*n,hipMemcpyHostToDevice);

	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
		
	if(t_f != NULL){
	free(t_f);
	t_f = NULL;
	}

	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
	//solve normlize
	hipComplex *d_v,*d_a;
	hipMalloc((void**)&d_v,sizeof(hipComplex)*m*n);
	hipMalloc((void**)&d_a,sizeof(hipComplex)*n);
	
	#pragma unroll
	for(int i=0;i<n;i++){

	  normlize(d_fftData+i*m,m,1,d_v+i*m,d_a+i);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
		}
	  
	}

	hipFree(d_fftData);
	
	//transform forward
	hipComplex* h_v = (hipComplex*)malloc(sizeof(hipComplex)*m*n);
	hipComplex* h_v2 = (hipComplex*)malloc(sizeof(hipComplex)*m*n);
	hipMemcpy(h_v2,d_v,sizeof(hipComplex)*m*n,hipMemcpyDeviceToHost);
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tube;j++){
		h_v[i*tube+j] = h_v2[j*bat+i];
	  }
	}
	hipMemcpy(d_v,h_v,sizeof(hipComplex)*m*n,hipMemcpyHostToDevice);

	if(h_v2){
	   free(h_v2);
	   h_v2 = NULL;
	}
	//d_v and d_a take ifft
	hipfftHandle iplan;
	if(hipfftPlan1d(&iplan,tube,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}

	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(iplan,d_v+i*tube,d_v+i*tube,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//d_a take ifft
	if(hipfftPlan1d(&iplan,tube,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	if(hipfftExecC2C(iplan,d_a,d_a,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipComplex* h_a = (hipComplex*)malloc(sizeof(hipComplex)*n);
	hipMemcpy(h_v,d_v,sizeof(hipComplex)*m*n,hipMemcpyDeviceToHost);
	hipMemcpy(h_a,d_a,sizeof(hipComplex)*n,hipMemcpyDeviceToHost);
	//transform inverse
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tube;j++){
		v[j*bat+i]=h_v[i*tube+j].x/tube;
	   }
	}
	
	for(int i=0;i<tube;i++){
	  a[i]=h_a[i].x/tube;
	}
	
	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(h_v != NULL){
	   free(h_v);
	   h_v = NULL;
	}
	if(h_a != NULL){
	   free(h_a);
	   h_a = NULL;
	}
	if(d_v != NULL){
	   hipFree(d_v);
	   d_v = NULL;
	}
	if(d_a != NULL){
	   hipFree(d_a);
	   d_a = NULL;
	}
}
