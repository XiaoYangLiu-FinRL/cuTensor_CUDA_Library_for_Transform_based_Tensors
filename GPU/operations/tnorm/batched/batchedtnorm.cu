#include "hip/hip_runtime.h"
#include "norm.h"
void batchedtnorm(float* t,const int m, const int n,const int tube,float* result){
   int num=m*n*tube;
   float* y;
   hipMalloc((void**)&y,num*sizeof(float));
   //set vector
   if(hipblasSetVector(num,sizeof(float),t,1,y,1) != HIPBLAS_STATUS_SUCCESS){
        fprintf(stdout,"[%s]:[%d] hipblasSnrm2 error!",__FUNCTION__,__LINE__);
        return;
   }
   //create cublasHandle;
   hipblasHandle_t handle;
   hipblasCreate(&handle);
   //norm
   if(hipblasSnrm2(handle,num,y,1,result) != HIPBLAS_STATUS_SUCCESS){
        fprintf(stdout,"[%s]:[%d] hipblasSnrm2 error!",__FUNCTION__,__LINE__);
        return;
   }
   
   if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS){
        fprintf(stdout,"[%s]:[%d] hipblasSnrm2 error!",__FUNCTION__,__LINE__);
        return;
   }
   if(y != NULL){
        hipFree(y);
        y=NULL;
   }    
}
