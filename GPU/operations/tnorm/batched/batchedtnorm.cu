#include "hip/hip_runtime.h"
#include "norm.h"
#include "based.h"
/**
* normlization of tnesor t:
*       t = v * a;
* ||v|| = 1,it's mean <v,v> = e.
* INPUT: t is m×1×n
*
* OUTPUT: v*a  = t.
*/
void batchedtnorm(float* t,const int m, const int n,float* v,float* a){
	int ht = n/2+1;
	int bat = m;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*n);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*m*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*n,hipMemcpyHostToDevice);	
	
	//tfft
	hipfftHandle plan;
	int n_f[1] = {n};
	int in[1] = {n};
	int ou[1] = {ht}; 
	int stride_in = bat,dist_in = 1;
	int stride_ou = bat,dist_ou = 1;
	
	if(hipfftPlanMany(&plan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_R2C,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	} 
	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
	
	//solve normlize
	
	hipComplex *d_hv,*d_ha;
	hipMalloc((void**)&d_hv,sizeof(hipComplex)*m*ht);
	hipMalloc((void**)&d_ha,sizeof(hipComplex)*ht);
	
	normlize(d_fftData,m,ht,d_hv,d_ha);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipFree(d_fftData);	
	
	//d_hv and d_ha take ifft 
	int threads = 0;
	int blocks = 0;	
	int num = 0;
	
	float *d_v,*d_a;
	hipMalloc((void**)&d_v,sizeof(float)*m*n);
	hipMalloc((void**)&d_a,sizeof(float)*n);
 
	hipfftHandle iplan;
	in[0] = ht;
	ou[0] = n;

	if(hipfftPlanMany(&iplan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_C2R,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	} 
	if(hipfftExecC2R(iplan,d_hv,d_v)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	num = m*n;
	if(num < 512){
	    threads = num;
	    blocks = 1;
	}else{
	    threads = 512;
	    blocks = ((num%512) == 0)?num/512:num/512+1;
	}

	fftResultProcess<<<blocks,threads>>>(d_v,num,n);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(v,d_v,sizeof(float)*m*n,hipMemcpyDeviceToHost);
	
	hipFree(d_hv);
	hipFree(d_v);
	
	stride_in = 1;
	stride_ou = 1;
	dist_in = 1;
	dist_ou = 1;
	bat = 1;

	if(hipfftPlanMany(&iplan,1,n_f,in,stride_in,dist_in,ou,stride_ou,dist_ou,
				HIPFFT_C2R,bat) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftExecC2R(iplan,d_ha,d_a)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
 	}	
	num = n;
	if(n < 512){
	    threads = num;
	    blocks = 1;
	}else{
	    threads = 512;
	    blocks = ((num%512) == 0)?num/512:num/512+1;
	}	

	fftResultProcess<<<blocks,threads>>>(d_a,num,n);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(a,d_a,sizeof(float)*n,hipMemcpyDeviceToHost);
	
	hipFree(d_ha);	
	hipFree(d_a);
}
