#include "fft.h"


void batchedTfft(float *t,int l,int bat,hipfftComplex *tf)
{
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    hipMemcpy(d_t,t,l*bat*sizeof(float),hipMemcpyHostToDevice);

    hipfftHandle plan;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {l};
    int on[1] = {hl};
    size_t worksize=0;

    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    //estimat of the work size
    hipfftGetSizeMany(plan,1,n,in,stride,dist,on,stride,dist,HIPFFT_R2C,bat,&worksize);
    printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));
    if
        (hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    
    hipDeviceSynchronize();
    hipMemcpy(tf,d_fftData,hl*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_t);
    hipFree(d_fftData);
}
void batchedTifft(float *t,int l,int bat,hipfftComplex *tf)
{
    //transform
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,tf,hl*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {hl};
    int on[1] = {l};
    
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUIFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecC2R(plan,(hipfftComplex*)d_fftData, d_t)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUIFFT error: EXEC  failed");
            return; 
        }
    hipDeviceSynchronize();
    hipMemcpy(t,d_t,l*bat*sizeof(float),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    hipFree(d_t);
    //transform
    for (int i=0; i<l*bat; i++)
        t[i] = t[i]/l;
}
