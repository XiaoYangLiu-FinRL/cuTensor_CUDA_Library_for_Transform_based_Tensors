#include "fft.h"
#include "based.h"


void batchedTfft(float *t,int l,int bat,hipfftComplex *tf)
{
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    hipMemcpy(d_t,t,l*bat*sizeof(float),hipMemcpyHostToDevice);

    hipfftHandle plan;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {l};
    int on[1] = {hl};
    size_t worksize=0;

    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    //estimat of the work size
    hipfftGetSizeMany(plan,1,n,in,stride,dist,on,stride,dist,HIPFFT_R2C,bat,&worksize);
    printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));
    if
        (hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    
    hipDeviceSynchronize();
    hipMemcpy(tf,d_fftData,hl*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_t);
    hipFree(d_fftData);
}
void batchedTifft(float *t,int l,int bat,hipfftComplex *tf)
{
    //transform
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,tf,hl*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {hl};
    int on[1] = {l};
    
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUIFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecC2R(plan,(hipfftComplex*)d_fftData, d_t)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUIFFT error: EXEC  failed");
            return; 
        }
    hipDeviceSynchronize();
       	int num=bat*l;
	int threads,blocks;
        if(num<512){
          threads=num;
          blocks=1;
        }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads>>>(d_t,num,l);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

    hipMemcpy(t,d_t,l*bat*sizeof(float),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    hipFree(d_t);
}
