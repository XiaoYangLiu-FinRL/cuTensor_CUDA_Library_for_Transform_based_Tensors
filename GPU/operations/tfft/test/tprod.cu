#include "tprod.h"
void tprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe) {
    int ht = tupe/2 + 1;//half tupe
    int bat1 = row*rank;
    int bat2 = col*rank;
    int bat = bat1 + bat2;
    float *d_t;
    hipMalloc((void**)&d_t, tupe*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,ht*bat*sizeof(hipfftComplex));
    hipMemcpy(d_t,t1,tupe*bat1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_t+tupe*bat1, t2,tupe*bat2*sizeof(float),hipMemcpyHostToDevice);
    //tfft

    hipfftHandle plan;
    int n[1] = {tupe};
    int stride = bat1, dist = 1;
    int in[1] = {tupe};
    int on[1] = {ht};
    
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat1)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    
    hipDeviceSynchronize();
    stride = bat2;
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat2)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecR2C(plan, d_t+tupe*bat1,d_fftData+ht*bat1)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    hipFree(d_t);
    //gemmbatched

    hipfftComplex* d_Tf;
    hipMalloc((void**)&d_Tf,ht*row*col*sizeof(hipfftComplex));
    hipblasHandle_t handle;
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;
    int Am = row;
    int An = rank;
    int Bn = col;
    int Bm = rank;
    int strA = Am*An;
    int strB = Bm*Bn;
    int strC = Am*Bn;
    hipblasCreate(&handle);
    hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
            &alpha, d_fftData, Am, strA, d_fftData+Am*Bm*ht, Bm, strB, &beta,
            d_Tf, Am, strC, ht);
    hipblasDestroy(handle);

    //Tifft

    in[0] = ht;
    on[0] = tupe;
    float* d_T;
    bat = row*col;
    stride = bat;
    hipMalloc((void**)&d_T, tupe*bat*sizeof(float));
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecC2R(plan,(hipfftComplex*)d_Tf, d_T)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    
    hipDeviceSynchronize();
    hipMemcpy(T,d_T,tupe*bat*sizeof(float),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    hipFree(d_Tf);
    hipFree(d_T);
//transform
    for (int i=0; i<tupe*bat; i++)
        T[i] = T[i]/tupe;

}
/*void streamedtprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe) {
    hipfftComplex *t1f = new hipfftComplex[row*rank*tupe];
    hipfftComplex *t2f = new hipfftComplex[rank*col*tupe];
    Tfft(t1,tupe,row*rank,t1f);
    Tfft(t2,tupe,rank*col,t2f);
    hipfftComplex *Tf = new hipfftComplex[row*col*tupe];
    gemmStrideStreamed(t1f, t2f, Tf, row, rank, tupe, col);
    delete[] t1f;
    delete[] t2f;
    Tifft(T,tupe,row*col,Tf);
    delete[] Tf;
}*/

