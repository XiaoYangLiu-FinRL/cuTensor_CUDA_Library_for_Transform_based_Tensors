#include "fft.h"


void Tfft(float *t,int l,int bat,hipfftComplex *tf)
{
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    // hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    // if (hipGetLastError() != hipSuccess){
    //   printf(stderr, "Cuda error: Failed to allocate\n");
    //   return; 
    // }

    hipMemcpy(d_t,t,l*bat*sizeof(float),hipMemcpyHostToDevice);

    hipfftHandle plan;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {l};
    int on[1] = {hl};
    
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    
    hipDeviceSynchronize();
    hipMemcpy(tf,d_fftData,hl*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_t);
    hipFree(d_fftData);
}
void Tifft(float *t,int l,int bat,hipfftComplex *tf)
{
  //to be update;use stream?
//transform
    int hl = l/2+1;
    float *d_t;
    hipMalloc((void**)&d_t, l*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,hl*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,tf,hl*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    int n[1] = {l};
    int stride = bat, dist = 1;
    int in[1] = {hl};
    int on[1] = {l};
    
    if
        (hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)!=HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return; 
        }
    if
        (hipfftExecC2R(plan,(hipfftComplex*)d_fftData, d_t)
         != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: EXEC  failed");
            return; 
        }
    hipDeviceSynchronize();
    hipMemcpy(t,d_t,l*bat*sizeof(float),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    hipFree(d_t);
//transform
    for (int i=0; i<l*bat; i++)
        t[i] = t[i]/l;
}
void streamedTfft(float *t,int l,int bat,hipfftComplex *tf) {
	hipfftComplex *t_f = new hipfftComplex[l*bat];
//transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j].x=t[j*bat+i];
        t_f[i*l+j].y=0;
      }
    hipfftComplex *d_fftData;
    // hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    // if (hipGetLastError() != hipSuccess){
    //   printf(stderr, "Cuda error: Failed to allocate\n");
    //   return; 
    // }

    hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    hipfftPlan1d(&plan,l,HIPFFT_C2C, 1);
    hipStream_t *streams = (hipStream_t *) malloc(bat*sizeof(hipStream_t));
    for (int i=0; i<bat; i++)
        hipStreamCreate(&streams[i]);
    for (int i=0; i<bat; i++) {
        hipfftSetStream(plan, streams[i]);
        hipfftExecC2C(plan,(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_FORWARD);
    }
    hipDeviceSynchronize();
    hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    free(streams);
    hipFree(d_fftData);
//transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            tf[j*bat+i]=t_f[i*l+j];
          }
    delete[] t_f;
}

void streamedTifft(float *t, int l, int bat, hipfftComplex *tf){
  hipfftComplex *t_f = new hipfftComplex[l*bat];
//transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j]=tf[j*bat+i];
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    hipfftPlan1d(&plan,l,HIPFFT_C2C,1);
    hipStream_t *streams = (hipStream_t *) malloc(bat*sizeof(hipStream_t));
    for (int i=0; i<bat; i++)
        hipStreamCreate(&streams[i]);
    for (int i=0; i<bat; i++) {
        hipfftSetStream(plan, streams[i]);
        hipfftExecC2C(plan,(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_BACKWARD);
        }
    hipDeviceSynchronize();
    hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    free(streams);
    hipFree(d_fftData);
//transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            t[j*bat+i]=t_f[i*l+j].x/l;
          }
    delete[] t_f;
}
