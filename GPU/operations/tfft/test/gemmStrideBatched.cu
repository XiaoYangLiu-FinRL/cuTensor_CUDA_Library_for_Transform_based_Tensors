#include "gemmStrideBatched.h"

void gemmStrideBatched(hipComplex *A, hipComplex *B, hipComplex *C, int Am, int An, int Ak,  int Bn) {
    hipblasHandle_t handle;
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;
    int Bm = An;
    int Bk = Ak;
    int strA = Am*An;
    int strB = Bm*Bn;
    int strC = Am*Bn;
    int batchCount = Ak;

    hipComplex *d_A, *d_B, *d_C;
    hipblasCreate(&handle);
    hipMalloc ((void**)&d_A, sizeof(hipComplex) * Am*An*Ak);
    hipMalloc ((void**)&d_B, sizeof(hipComplex) * Bm*Bn*Bk);
    hipMalloc ((void**)&d_C, sizeof(hipComplex) * Am*Bn*Ak);

    hipMemcpy(d_A, A, sizeof(hipComplex) * Am*An*Ak, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(hipComplex) * Bm*Bn*Bk, hipMemcpyHostToDevice);

    hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm, &alpha, d_A, Am, strA, d_B, Bm, strB, &beta, d_C, Am, strC, batchCount);

    hipblasDestroy(handle);
    hipMemcpy(C, d_C, sizeof(hipComplex) * Am*Bn*Ak, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

