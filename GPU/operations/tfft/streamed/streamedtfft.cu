#include "fft.h"
void streamedTfft(float *t,int l,int bat,hipfftComplex *tf) {
    hipfftComplex *t_f;
    if( hipHostAlloc((void**)&t_f,sizeof(hipfftComplex)*l*bat,hipHostMallocDefault) != hipSuccess){
        	fprintf(stdout,"[%s]:[%d] hipHostAlloc error!",__FUNCTION__,__LINE__);
        	return;	
    }
    //transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j].x=t[j*bat+i];
        t_f[i*l+j].y=0;
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));

    //set stream
    hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t)*MAX_PLAN1D_SIZE);
    hipfftHandle* plan = (hipfftHandle*)malloc(sizeof(hipfftHandle)*MAX_PLAN1D_SIZE);
    memset(plan,0,sizeof(hipfftHandle));
    
    int bat_num=bat/MAX_PLAN1D_SIZE;
    int s_bat=bat%MAX_PLAN1D_SIZE;  
//  
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
       if( hipStreamCreate(&stream[i]) != hipSuccess){
        	fprintf(stdout,"[%s]:[%d] create stream error!",__FUNCTION__,__LINE__);
        	return;	
       }
       }
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
        if(hipfftPlan1d(&plan[i],l,HIPFFT_C2C, 1)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft hipfftPlan1d error!",__FUNCTION__,__LINE__);
        	return;	
    	}
       if(hipfftSetStream(plan[i], stream[i]) != HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft set stream error!",__FUNCTION__,__LINE__);
        	return;	
       }
    }
   if(bat_num > 0){
  //  int j=0;
    for(int j=0;j<bat_num;j++){
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
       if( hipMemcpyAsync(d_fftData+i*l+j*MAX_PLAN1D_SIZE*l,t_f+i*l+j*MAX_PLAN1D_SIZE*l,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }
        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l+j*MAX_PLAN1D_SIZE*l),(hipfftComplex*)(d_fftData+i*l+j*MAX_PLAN1D_SIZE*l),HIPFFT_FORWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

        hipMemcpyAsync(t_f+i*l+j*MAX_PLAN1D_SIZE*l,d_fftData+i*l+j*MAX_PLAN1D_SIZE*l,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);

    }
    }

    
    #pragma unroll
    for(int i=0;i<s_bat;i++){
         if( hipMemcpyAsync(d_fftData+i*l+bat_num*MAX_PLAN1D_SIZE*l,t_f+i*l+bat_num*MAX_PLAN1D_SIZE*l,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }

        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l+bat_num*MAX_PLAN1D_SIZE*l),(hipfftComplex*)(d_fftData+i*l+bat_num*MAX_PLAN1D_SIZE*l),HIPFFT_FORWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

         hipMemcpyAsync(t_f+i*l+bat_num*MAX_PLAN1D_SIZE*l,d_fftData+i*l+bat_num*MAX_PLAN1D_SIZE*l,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);
         }

    }else{
    #pragma unroll
    for(int i=0;i<s_bat;i++){
         if( hipMemcpyAsync(d_fftData+i*l,t_f+i*l,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }

        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_FORWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

         hipMemcpyAsync(t_f+i*l,d_fftData+i*l,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);
         }
    }
    //synchronize stream

    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
    hipStreamSynchronize(stream[i]);
    }

    //destroy stream
   
   #pragma unroll
    for (int i=0; i<MAX_PLAN1D_SIZE; i++){
        hipfftDestroy(plan[i]);
        hipStreamDestroy(stream[i]);
    }
        free(plan);
        free(stream);
        hipFree(d_fftData);
    
    //transform

    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            tf[j*bat+i]=t_f[i*l+j];
          }
    hipHostFree(t_f);
}

void streamedTifft(float *t, int l, int bat, hipfftComplex *tf){
    hipfftComplex *t_f;
    hipHostAlloc((void**)&t_f,l*bat*sizeof(hipfftComplex),hipHostMallocDefault);
    //transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j]=tf[j*bat+i];
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    
    hipStream_t * stream = (hipStream_t *)malloc(MAX_PLAN1D_SIZE*sizeof(hipStream_t));
    hipfftHandle* plan = (hipfftHandle*)malloc(MAX_PLAN1D_SIZE*sizeof(hipfftHandle));
    memset(plan,0,sizeof(hipfftHandle)); 

    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }
    #pragma unroll
   for(int i=0;i<MAX_PLAN1D_SIZE;i++){
        if(hipfftPlan1d(&plan[i],l,HIPFFT_C2C,1)!=HIPFFT_SUCCESS){
    	fprintf(stdout,"[%s]:[%d] fft hipfftPlan1d error!",__FUNCTION__,__LINE__);
    	return;	
    	}
       if( hipfftSetStream(plan[i],stream[i]) != HIPFFT_SUCCESS){
    	fprintf(stdout,"[%s]:[%d] fft set stream error!",__FUNCTION__,__LINE__);
    	return;	
       }
    }
    int bat_num=bat/MAX_PLAN1D_SIZE;
    int s_bat=bat%MAX_PLAN1D_SIZE;  
    
    if(bat_num !=0){
    for(int j=0;j<bat_num;j++){
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
         if( hipMemcpyAsync(d_fftData+i*l+j*l*MAX_PLAN1D_SIZE,t_f+i*l+j*l*MAX_PLAN1D_SIZE,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }

        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l+j*l*MAX_PLAN1D_SIZE),(hipfftComplex*)(d_fftData+i*l+j*l*MAX_PLAN1D_SIZE),HIPFFT_BACKWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

         hipMemcpyAsync(t_f+i*l+j*l*MAX_PLAN1D_SIZE,d_fftData+i*l+j*l*MAX_PLAN1D_SIZE,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);
    }
    }
    #pragma unroll
    for(int i=0;i<s_bat;i++){
         if( hipMemcpyAsync(d_fftData+i*l+bat_num*l*MAX_PLAN1D_SIZE,t_f+i*l+bat_num*l*MAX_PLAN1D_SIZE,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }

        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l+bat_num*l*MAX_PLAN1D_SIZE),(hipfftComplex*)(d_fftData+i*l+MAX_PLAN1D_SIZE*l*bat_num),HIPFFT_BACKWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

         hipMemcpyAsync(t_f+i*l+bat_num*MAX_PLAN1D_SIZE*l,d_fftData+i*l+bat_num*MAX_PLAN1D_SIZE*l,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);
         }

    }else{
    #pragma unroll
    for(int i=0;i<s_bat;i++){
         if( hipMemcpyAsync(d_fftData+i*l,t_f+i*l,l*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
            fprintf(stdout,"[%s]:[%d] cudaMencpyAsync error!",__FUNCTION__,__LINE__);
            return;
        }

        if(hipfftExecC2C(plan[i],(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_BACKWARD)!=HIPFFT_SUCCESS){
        	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
         	return;
    	}

         hipMemcpyAsync(t_f+i*l,d_fftData+i*l,l*sizeof(hipfftComplex),hipMemcpyDeviceToHost,stream[i]);
         }
    }
    //synchronize stream
   
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    #pragma unroll
    for(int i=0;i<MAX_PLAN1D_SIZE;i++){
       hipfftDestroy(plan[i]);
       hipStreamDestroy(stream[i]);
    }
    free(plan);
    free(stream);
    hipFree(d_fftData);
    //transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            t[j*bat+i]=t_f[i*l+j].x/l;
          }
    hipHostFree(t_f);
}
