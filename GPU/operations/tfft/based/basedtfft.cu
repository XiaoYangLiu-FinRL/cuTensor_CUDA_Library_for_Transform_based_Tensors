#include "fft.h"
void basedTfft(float *t,int l,int bat,hipfftComplex *tf) {
    hipfftComplex *t_f = new hipfftComplex[l*bat];
    //transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j].x=t[j*bat+i];
        t_f[i*l+j].y=0;
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));

    hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    if(hipfftPlan1d(&plan,l,HIPFFT_C2C, 1)!=HIPFFT_SUCCESS){
	fprintf(stdout,"[%s]:[%d] fft hipfftPlan1d error!",__FUNCTION__,__LINE__);
	return;	
	}
     
    for (int i=0; i<bat; i++) {
   
        if(hipfftExecC2C(plan,(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_FORWARD)!=HIPFFT_SUCCESS){
	fprintf(stdout,"[%s]:[%d] fft cufftExecC2c error!",__FUNCTION__,__LINE__);
	return;
	}
    }

    hipDeviceSynchronize();
    hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    //transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            tf[j*bat+i]=t_f[i*l+j];
          }
    delete[] t_f;
}

void basedTifft(float *t, int l, int bat, hipfftComplex *tf){
    hipfftComplex *t_f = new hipfftComplex[l*bat];
    //transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j]=tf[j*bat+i];
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
     if(hipfftPlan1d(&plan,l,HIPFFT_C2C,1)!=HIPFFT_SUCCESS){
	fprintf(stdout,"[%s]:[%d] ifft hipfftPlan1d error!",__FUNCTION__,__LINE__);
	return;
	}
   
    for (int i=0; i<bat; i++) {
        if(hipfftExecC2C(plan,(hipfftComplex*)(d_fftData+i*l),(hipfftComplex*)(d_fftData+i*l),HIPFFT_BACKWARD)!=HIPFFT_SUCCESS){
	fprintf(stdout,"[%s]:[%d] ifft cufftExecC2c error!",__FUNCTION__,__LINE__);
	return;
	}
        }

    hipDeviceSynchronize();
    hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
    //transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            t[j*bat+i]=t_f[i*l+j].x/l;
          }
    delete[] t_f;
}
