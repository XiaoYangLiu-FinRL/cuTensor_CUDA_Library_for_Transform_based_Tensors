#include "hip/hip_runtime.h"
#include "based.h"

__global__ void float2cuComplex(const float* d_in,const int length,hipComplex* d_out){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < length){
	d_out[tid].x = d_in[tid];
	__syncthreads();
	d_out[tid].y = 0;
	}
}

__global__ void cuComplex2float(const hipComplex* d_in,const int length,float* d_out){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < length){
	d_out[tid] = d_in[tid].x;
	}
}
#if 0
__global__ void d_float2cuComplex(const float* d_in,const int length,hipComplex* d_out){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < length){
	d_out[tid].x = d_in[tid];
	__syncthreads();
	d_out[tid].y = 0;
	}
}

__global__ void d_cuComplex2float(const hipComplex* d_in,const int length,float* d_out){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < length){
	d_out[tid] = d_in[tid].x;
	}
}
void float2Complex(const float* d_in, const int length,hipComplex* d_out){
	int threads = 0;
	int blocks = 0;
	if(length < 512){
`	    threads = length;
	    blocks =  1;
	}else{
	    threads = 512;
	    blocks = ((length%512) == 0)?length/512:length/512+1;
	}
	d_float2cuComplex<<<blocks,threads>>>(d_in,length,d_out);
}

void cuComplex2float(const hipComplex* d_in,const int length,float* d_out){
	int threads = 0;
	int blocks = 0;
	if(length < 512){
	   threads = length;
	   blocks = 1;
	}else{
	   threads = 512;
	   blocks = ((length%512) == 0)?length/512:length/512+1;
	}
	d_cuComplex2float<<<blocks,threads>>>(d_in,length,d_out);
}
#endif
