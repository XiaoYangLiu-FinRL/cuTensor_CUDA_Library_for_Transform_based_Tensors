#include "hip/hip_runtime.h"
#include "based.h"
__global__ void float2cuComplex(float* d_in,const int length,hipComplex* d_out){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < length){
	d_out[tid].x = d_in[tid];
	__syncthreads();
	d_out[tid].y = 0;
	}
}
