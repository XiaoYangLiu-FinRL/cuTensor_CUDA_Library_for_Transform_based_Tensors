#include "hip/hip_runtime.h"
/**
 * @device matview_transopse
 * Create on:Nov 28 2018
 * @author: haili
 * the size of tensor is m×n×batch
 */
#include "based.h"
__global__ void d_batch_c_transpose(hipComplex* A,const int m,const int n,const int batch,hipComplex* T){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*batch){
		T[(tid/(m*n))*n*m+(tid%(m*n))/n+((tid%(m*n))%n)*m].x=A[tid].x;
		T[(tid/(m*n))*n*m+(tid%(m*n))/n+((tid%(m*n))%n)*m].y=0-A[tid].y;
		tid+=t_n;
	}
	__syncthreads();
}
__global__ void d_batch_f_transpose(float* A,const int m,const int n,const int batch,float* T){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*batch){
		T[(tid/(m*n))*n*m+(tid%(m*n))/n+((tid%(m*n))%n)*m]=A[tid];
		tid+=t_n;
	}
	__syncthreads();
}

void batchedctranspose(hipComplex* A,const int m,const int n,const int batch,hipComplex* T){
    int threads;
    int blocks;
    int num= m * n * batch; 
    if(num < 512){
        threads=num;
        blocks=1;
    }else{
        threads=512;
        blocks= (num%512 ==0)?num/512:num/512+1;
    }

    d_batch_c_transpose<<<blocks,threads>>>(A,m,n,batch,T);
}
void batchedftranspose(float* A,const int m,const int n,const int batch,float* T){
    int threads;
    int blocks;
    int num= m * n * batch; 
    if(num < 512){
        threads=num;
        blocks=1;
    }else{
        threads=512;
        blocks= (num%512 ==0)?num/512:num/512+1;
    }

    d_batch_f_transpose<<<blocks,threads>>>(A,m,n,batch,T);
}
