#include "hip/hip_runtime.h"
#include "based.h"
__global__ void conMatrixK(hipfftComplex* d_fftData, float* d_k,const int m, const int n,const int batch){
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int num = m*n;
	int id;
	int tid;

	if(tidx < m*n*batch*4){

	id = tidx/(m*n*4);
	tid = tidx%(m*n*4);

	if(tid < num){
		d_k[tid/m*2*m+tid%m +id*4*m*n] = d_fftData[tid+id*m*n].x;
	}
	
	if(num <= tid < 2*num){
		d_k[(tid%num)/m*2*m+(tid%num)%m+m+id*4*m*n] = -d_fftData[tid%num+id*m*n].y;
	}

	if(2*num <= tid < 3*num){
		d_k[(tid%num)/m*2*m+(tid%num)%m+2*m*n +id*4*m*n] = d_fftData[tid%num+id*m*n].y;
		
	}
	
	if(3*num <= tid < 4*num){
		d_k[(tid%num)/m*2*m+(tid%num)%m+m+2*m*n +id*4*m*n] = d_fftData[tid%num+id*m*n].x;
		
	}

	}
	__syncthreads();
}
