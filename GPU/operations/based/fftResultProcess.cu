#include "hip/hip_runtime.h"
#include "based.h"
__global__ void fftResultProcess(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}
