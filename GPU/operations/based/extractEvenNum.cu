#include "hip/hip_runtime.h"
#include"based.h"
/**
*INPUT:    d_k of size 4*m*n * batch
*OUTPUT:   d_U of size m*n * batch
*K matrix row is 2*m,col is 2*n,batch is the number of K matrix.
*U matrix row is m,col is n, batch is the number of U matrix.
*/
__global__ void extractEvenNumU(float* d_k, hipComplex* d_U, const int m,const int n, const int batch){
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int id,sid;
	int tid,stid;
	
	if(tidx< 2*m*n*batch){

	id = tidx/(2*m*n);	
	tid = tidx%(2*m*n);

	sid = tid/(2*m);
	stid = tid%(2*m);
	if(stid < m){
		d_U[stid + sid*m +id*m*n].x=d_k[ stid + sid*4*m + id*m*n*4];
	}

	if(m <= stid < 2*m){
		d_U[stid%m + sid*m +id*m*n].y=d_k[ stid%m + m +sid*4*m + id*m*n*4];
	}
	}	
	__syncthreads();
}
/**
*INPUT: d_s of size 2m*2n * batch
*OUTPUT: ds_extract of size m* n * batch
*/
__global__ void extractEvenNumS(float* d_s, float* ds_extract, const int m,const int n, const int batch){
	const int tidx = blockIdx.x * blockDim.x +threadIdx.x;
	int min = ((2*m<2*n)?2*m:2*n);
	int id;
	int tid;
	if(tidx < batch*(min/2)){
		id = tidx/(min/2);
		tid = tidx%(min/2);
		ds_extract[tid + id*(min/2)]=d_s[tid*2 + id*min];
	}	
	__syncthreads();
}
