#include "hip/hip_runtime.h"
#include"based.h"
__global__ void CopyUpperSubmatrix( const hipComplex* d_in,hipComplex* d_out,
		const int M, const int N, const int subM){
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	if( i < subM*N)
	d_out[(i/subM)*subM+i%subM] = d_in[(i/subM)*M+i%subM];
}

void qrsolve(hipComplex* d_A,hipComplex* d_B,const int m,const int n,const int k,hipComplex* d_X){

const int M = m;
const int N = n;
const int K = k;
const int min = Min(m,n);

//define handles
hipsolverHandle_t cusolverH = NULL;
hipblasHandle_t cublasH = NULL;

//create handles
if( hipsolverDnCreate( &cusolverH ) != HIPSOLVER_STATUS_SUCCESS ){
	fprintf(stdout,"[%s]:[%d] hipsolverDnCreate error!",__FUNCTION__,__LINE__);
	return;
}
if( hipblasCreate( &cublasH ) != HIPBLAS_STATUS_SUCCESS ){
	fprintf(stderr,"[%s]:[%d] hipblasCreate error!",__FUNCTION__,__LINE__);
	return;
}
hipComplex  *d_work, *d_work2, *d_tau;
int *d_devInfo, devInfo;
hipMalloc( (void**)&d_tau,sizeof(hipComplex)* min);
hipMalloc( (void**)&d_devInfo, sizeof(int));
int bufsize,bufsize2;

// in-place A=QR
if( hipsolverDnCgeqrf_bufferSize(
		cusolverH,
		M,
		N,
		d_A,
		M,
		&bufsize
		) != HIPSOLVER_STATUS_SUCCESS ){
	fprintf(stdout,"[%s]:[%d] hipsolverDnCgeqrf_bufferSize error!",__FUNCTION__,__LINE__);
	return;
}

hipMalloc( (void**)&d_work, sizeof(hipComplex)* bufsize);

if( hipsolverDnCgeqrf(
		cusolverH,
		M,
		N,
		d_A,
		M,
		d_tau,
		d_work,
		bufsize,
		d_devInfo
		) != HIPSOLVER_STATUS_SUCCESS ){
	fprintf(stdout,"[%s]:[%d] hipsolverDnCgeqrf error!",__FUNCTION__,__LINE__);
	hipMemcpy(&devInfo, d_devInfo, sizeof(int), hipMemcpyDeviceToHost);
	printf("Info:%d\n",devInfo);
	return;
}
hipDeviceSynchronize();

if( d_work ) hipFree(d_work);
//Q`*B
if( hipsolverDnCunmqr_bufferSize(
		cusolverH,
		HIPBLAS_SIDE_LEFT,
		HIPBLAS_OP_C,
		M,
		K,
		min,
		d_A,
		M,
		d_tau,
		d_B,
		M,
		&bufsize2
	) != HIPSOLVER_STATUS_SUCCESS){
	fprintf(stdout,"[%s]:[%d] cusolverDnCunmqr_buffersize error!",__FUNCTION__,__LINE__);
	return;
}

if (hipMalloc((void**)&d_work2, sizeof(hipComplex)* bufsize2) != hipSuccess){
	fprintf(stdout,"[%s]:[%d] cuda runtime API error!",__FUNCTION__,__LINE__);
	return;
}

if( hipsolverDnCunmqr(
		cusolverH,
		HIPBLAS_SIDE_LEFT,
		HIPBLAS_OP_C,
		M,
		K,
		min,
		d_A,
		M,
		d_tau,
		d_B,
		M,
		d_work2,
		bufsize2,
		d_devInfo)
	!= HIPSOLVER_STATUS_SUCCESS){
	fprintf(stdout,"[%s]:[%d] hipsolverDnCunmqr error!",__FUNCTION__,__LINE__);
	hipMemcpy(&devInfo, d_devInfo, sizeof(int), hipMemcpyDeviceToHost);
	printf("Info:%d\n",devInfo);
	return;
}
hipDeviceSynchronize();

if( d_work2 ) hipFree(d_work2);
if(d_tau) hipFree(d_tau);

if(d_devInfo) hipFree(d_devInfo);

hipComplex *d_R;
hipMalloc((void**)&d_R, sizeof(hipComplex)* min * N);
int threads=0;
int blocks=0;
int num = min*N;
if(num<512){
	threads=num;
	blocks=1;
}else{
	threads=512;
	blocks = ((num%512) == 0)?num/512:num/512+1;
}
CopyUpperSubmatrix<<<blocks,threads>>>(d_A, d_R, M, N, min);

hipDeviceSynchronize();
num = min*K;
if(num<512){
	threads=num;
	blocks=1;
}else{
	threads=512;
	blocks = ((num%512) == 0)?num/512:num/512+1;
}
CopyUpperSubmatrix<<<blocks,threads>>>(d_B, d_X, M, K, min);
hipDeviceSynchronize();

//solve x = R \ (Q`*B)
hipComplex alphat;
alphat.x = 1;
alphat.y = 0;

if( hipblasCtrsm(
		cublasH,
		HIPBLAS_SIDE_LEFT,
		HIPBLAS_FILL_MODE_UPPER,
		HIPBLAS_OP_N,
		HIPBLAS_DIAG_NON_UNIT,
		min,
		K,
		&alphat,
		d_R,
		N,
		d_X,
		N
	) != HIPBLAS_STATUS_SUCCESS){
	fprintf(stdout,"[%s]:[%d] cusolverDnCunmqr_buffersize error!",__FUNCTION__,__LINE__);
	return;
}

hipDeviceSynchronize();

if(d_R) hipFree(d_R);

//Destroy handles
if( hipsolverDnDestroy( cusolverH) != HIPSOLVER_STATUS_SUCCESS ){
	fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy error!",__FUNCTION__,__LINE__);
	return;
}
if( hipblasDestroy( cublasH ) != HIPBLAS_STATUS_SUCCESS ){
	fprintf(stderr,"[%s]:[%d] hipblasDestroy error!",__FUNCTION__,__LINE__);
	return;
}
}

