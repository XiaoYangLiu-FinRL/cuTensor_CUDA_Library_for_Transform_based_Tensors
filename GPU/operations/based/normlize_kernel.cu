#include "hip/hip_runtime.h"
#include "based.h"
__global__ void d_normlize(hipComplex* d_in,const int len,hipComplex* v,hipComplex* a){
	int tidx = threadIdx.x;
	const int bidx = blockIdx.x;
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	int t_n = blockDim.x;
	__shared__ float temp_real[512];
	__shared__ float temp_image[512];
	float t_real = 0.0;
	float t_image = 0.0;
	while(tidx < len){
		t_real += d_in[tidx+bidx*len].x * d_in[tidx+bidx*len].x;
		t_image += d_in[tidx+bidx*len].y * d_in[tidx+bidx*len].y;
		 tidx += t_n;
	} 
	tidx = threadIdx.x;	
	temp_real[tidx] = t_real;
	temp_image[tidx] = t_image;
	__syncthreads();
	
	int i = 512/2;
	while(i != 0){
		if(tidx < i){
		  temp_real[tidx] += temp_real[tidx+i];
		  temp_image[tidx] += temp_image[tidx+i];
		}
		i /= 2;
	}
	if(tidx == 0){
	   temp_real[0]+=temp_image[0];
	}
	__syncthreads();
	
	if(temp_real[0] < 1.e-100){
		if(tidx == 0)
		{
		temp_real[0]= 2*len;
		}
	__syncthreads();
		while(tidx < len){
		   d_in[ tidx + bidx*len ].x = 1;
		   d_in[ tidx + bidx*len ].y = 1;
		   tidx += t_n;
		}
		tidx = threadIdx.x;
	__syncthreads();
	}
	
	if( tidx == 0){
		a[bidx].x = sqrt(temp_real[0]);
		a[bidx].y = 0;
	}
	__syncthreads();
	
	while( tidx < len){
		v[tidx + bidx*len].x = d_in[tidx + bidx*len].x/a[bidx].x;
		v[tidx + bidx*len].y = d_in[tidx + bidx*len].y/a[bidx].x;
		tidx += t_n;
	}
}

void normlize(hipComplex* d_in,const int len,const int batch,hipComplex* d_v,hipComplex* d_a){
	int threads = 512;
	int blocks = batch;
	d_normlize<<<blocks,threads>>>(d_in,len,d_v,d_a);
}
