#include "hip/hip_runtime.h"
#include "based.h"
/**
* note the array of result need to set 0;
*/
__global__ void d_batch_diag_mat(float* t,const int m,const int batch ,float* result){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int t_n=gridDim.x*blockDim.x;
    while(tid < m*m*batch){
	result[tid]=0;
	tid +=t_n;
    }
    __syncthreads();

    tid=blockDim.x*blockIdx.x+threadIdx.x;
    if(tid < m*batch){
        result[tid/m*(m*m)+tid%m*(m+1)]=t[tid];
    }
    __syncthreads();
}

void batcheddiagmat(float *t,const int m,const int batch,float* result){
    int threads;
    int blocks;
    int num = m*batch;
    if(num < 512){
        threads = num;
        blocks = 1;
    }else{
        threads = 512;
        blocks = (num%512 == 0)?num/512:num/512+1;
    }

    d_batch_diag_mat<<<blocks,threads>>>(t,m,batch,result);
}
