#include "hip/hip_runtime.h"
#include"based.h"
/**
*INPUT: d_hu of size m*n*(batch/2+1).
*OUTPUT: d_u of size m*n*batch.
*batch is length of d_u
*/
__global__ void  d_symmetricRecoverU(hipComplex* d_hu,const int m,const int n,const int batch,hipComplex* d_u){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	int ht = batch/2+1;
	int num = m*n*ht;
	if( tid < num){
	
	int id_mn = tid/(m*n);
	int ids_mn = tid%(m*n);

	d_u[ids_mn + id_mn*m*n] = d_hu[ids_mn+ id_mn*m*n];	
//	d_u[tid] = d_hu[tid];
	__syncthreads();

	if( (batch%2) == 1){
	   if( 0 < id_mn < ht){
		d_u[ (ht-id_mn-1)*m*n +ids_mn + ht*m*n].x=d_hu[ id_mn*m*n + ids_mn].x;
		__syncthreads();
		d_u[ (ht-id_mn-1)*m*n +ids_mn + ht*m*n].y=0-d_hu[ id_mn*m*n + ids_mn].y;
		}
	}
	__syncthreads();
	
	if( (batch%2) == 0){
		if( 0 < id_mn < (ht-1)){
		d_u[(ht-id_mn-2)*m*n +ids_mn + ht*m*n].x=d_hu[ id_mn*m*n + ids_mn ].x;
		__syncthreads();
		d_u[(ht-id_mn-2)*m*n +ids_mn + ht*m*n].y=0-d_hu[ id_mn*m*n + ids_mn ].y;
		}
	}
	__syncthreads();

	}
}

void symmetricRecoverU(hipComplex* d_hu,const int m,const int n,const int batch,hipComplex* d_u){
	int threads = 0;
	int blocks = 0;
	int ht = batch/2+1;
	int num =m*n*ht;
	if( num < 512){
	   threads = num;
	   blocks = 1;
	}else{
	   threads = 512;
	   blocks = ((num%512) == 0)?num/512:num/512+1;
	}
	d_symmetricRecoverU<<<blocks,threads>>>(d_hu,m,n,batch,d_u);
}
