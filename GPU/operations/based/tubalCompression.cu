#include "hip/hip_runtime.h"
#include "based.h"
/**
  t:the data of input.
  length:  each signal lengeth.
  batch:   the number of signal
  k :   the ratio of tubal.
*/
__global__ void d_tubal_compression(float* t,const int length,const int batch,const int k){
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int l = length - k;
	int num = k*batch;
	if( tid < num ){
	    int tid_l = tid/k;
	    int tid_l_s = tid%k;
	    t[tid_l*length+l+tid_l_s]=0;
	}
	__syncthreads();
}

void tubalCompression(float* t,const int length,const int batch, const int k){
	int threads = 0;
	int blocks = 0;
	int num = k*batch;
	if(num < 512){
	    threads = num;
	    blocks = 1;
	}else{
	    threads = 512;
	    blocks = ((num%512) == 0)?num/512:num/512+1;
	}
	
	d_tubal_compression<<<blocks,threads>>>(t,length,batch,k);
}
