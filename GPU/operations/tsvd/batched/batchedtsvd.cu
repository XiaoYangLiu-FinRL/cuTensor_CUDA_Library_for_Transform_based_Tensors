#include "svd.h"
#include "based.h"
#define KBLAS_Success 1

void batchedtsvd(float* t,const int m,const int n, const int tupe, hipComplex* host_u,float* S){
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
/*	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));
*/
	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
	//construct matrix K

//	hipComplex* h_fft=(hipComplex*)malloc(sizeof(hipComplex)*m*n*ht);
//	hipMemcpy(h_fft, d_fftData, sizeof(hipComplex)*m*n*ht,hipMemcpyDeviceToHost);
//	for(int i=0;i<m*n*ht;i++){
//	printf("h_fft %f	%f \n",h_fft[i].x,h_fft[i].y);
//	}

	float* d_k;
	hipMalloc((void**)&d_k,sizeof(float)*m*n*ht*4);
	
	int threads;
	int blocks;
	int num= (m*n*ht*4);
	if(num < 512){
	 threads = num;
	 blocks = 1;
	}else{
	 threads = 512;
	 blocks = ((num%512 ==0)?num/512:num/512+1);
	}

	conMatrixK<<<blocks,threads>>>(d_fftData,d_k,m,n,ht);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_fftData);	

//	float* h_k=(float*)malloc(sizeof(float)*4*m*n*ht);
//	hipMemcpy(h_k, d_k, sizeof(float)*4*m*n*ht,hipMemcpyDeviceToHost);
//	for(int i=0;i<4*m*n*ht;i++){
//	printf("h_k %f \n",h_k[i]);
//	}
			
	//tsvd
	int M=2*m;
	int N=2*n; 
	
	kblasHandle_t handle;
	kblasCreate( &handle );
	kblasSgesvj_batch_wsquery(handle, M, N, ht);

	if(kblasAllocateWorkspace(handle) != 1){
		fprintf(stdout,"[%s]:[%d] kblas  wsquery err!",__FUNCTION__,__LINE__);
		return;
	}

	int stride_s=((M<N)?M:N);
	float* d_s;
	hipMalloc((void**)&d_s, sizeof(float)*ht*stride_s);
	if(kblasSgesvj_batch_strided(handle, M, N, d_k, M, M*N, d_s, stride_s, ht) != 1){
		fprintf(stdout,"[%s]:[%d] kblas  svd  err!",__FUNCTION__,__LINE__);
		return;
	}

	
	kblasFreeWorkspace(handle);
	
//	hipMemcpy(h_k, d_k, sizeof(float)*4*m*n*ht,hipMemcpyDeviceToHost);
//	for(int i=0;i<4*m*n*ht;i++){
//	printf("take_h_k %f \n",h_k[i]);
//	}
	
	//extract elements 

	hipComplex* d_hu,*d_u;
	hipMalloc((void**)&d_hu,sizeof(hipComplex)*m*n*ht);
	hipMalloc((void**)&d_u,sizeof(hipComplex)*m*n*tupe);
	num = 4*m*n*ht;
	if(num < 512){
	 threads = num;
	 blocks = 1;
	}else{
	 threads = 512;
	 blocks = ((num%512 ==0)?num/512:num/512+1);
	}
	extractEvenNumU<<<blocks,threads>>>(d_k,d_hu,m,n,ht);	
	
//	printf("\n++++++++++++++++++++++++++\n");
//	hipComplex* h_u = (hipComplex*)malloc(sizeof(hipComplex)*m*n*ht);
//	hipMemcpy(h_u, d_hu, sizeof(hipComplex)*m*n*ht,hipMemcpyDeviceToHost);
//	for(int i=0;i<m*n*ht;i++){
//	printf("take_h_u %f	%f \n",h_u[i].x,h_u[i].y);
//	}
//	printf("\n++++++++++++++++++++++++++\n");

	//symmtricRecoverU

	symmetricRecoverU(d_hu,m,n,tupe,d_u);
	hipDeviceSynchronize();
//	printf("\n++++++++++++++++++++++++++\n");
//	hipComplex* hh_u = (hipComplex*)malloc(sizeof(hipComplex)*m*n*tupe);
//	hipMemcpy(hh_u, d_u, sizeof(hipComplex)*m*n*tupe,hipMemcpyDeviceToHost);
//	for(int i=0;i<m*n*tupe;i++){
//	printf("_sstake_h_u %f	%f \n",hh_u[i].x,hh_u[i].y);
//	}
//	printf("\n++++++++++++++++++++++++++\n");

	hipFree(d_hu);
	hipFree(d_k);
	//extract s
	float* ds_extract;
	hipMalloc((void**)&ds_extract,sizeof(float)*ht*((m<n)?m:n));	

	num = ht*((m<n)?m:n);
	if(num < 512){
	 threads = num;
	 blocks = 1;
	}else{
	 threads = 512;
	 blocks = ((num%512 ==0)?num/512:num/512+1);
	}
	extractEvenNumS<<<blocks,threads>>>(d_s,ds_extract,m,n,ht);
	
//	float* h_s2=(float*)malloc(sizeof(float)*num);
//	hipMemcpy(h_s2, ds_extract, sizeof(float)*num,hipMemcpyDeviceToHost);
//	for(int i=0;i<num;i++){
//	printf("take_h_s2 %f \n",h_s2[i]);
//	}

	hipFree(d_s);	
	//itfft_u

	//set stream
	hipStream_t stream[2];
	
	#pragma unroll
	for(int i=0;i<2;i++){
		hipStreamCreate(&stream[i]);
	}

	hipComplex* du;
	hipMalloc((void**)&du,sizeof(hipComplex)*m*n*tupe);

	hipfftHandle iplan =0;
	in[0] = tupe;
	on[0] = tupe;
	int stride_in = 1;
	int dist_in = tupe; 
	bat = m*n;
	stride = bat;
	
	hipfftSetStream(iplan,stream[0]);
	if (hipfftPlanMany(&iplan,1,n_f,in,stride_in,dist_in,on,stride,dist,
				HIPFFT_C2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
//	if(hipDeviceSynchronize() != hipSuccess){
//		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
//		return;
//	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride_in,dist_in,on,stride,dist,
			HIPFFT_C2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	//printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecC2C(iplan,(hipfftComplex*)d_u,du,HIPFFT_BACKWARD)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

//hipComplex* hhu = (hipComplex*)malloc(sizeof(hipComplex)*bat*tupe);
//hipMemcpy(hhu,du,sizeof(hipComplex)*bat*tupe,hipMemcpyDeviceToHost);
//printf("hhu_______________________________/n");
//for(int i=0;i<bat*tupe;i++){
//printf("[%f %f]	",hhu[i].x,hhu[i].y);
//}
//printf("hhu_______________________________/n");

	num=bat*tupe;
        if(num<512){
          threads=num;
          blocks=1;
        }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads,0,stream[0]>>>(du,num,tupe);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	hipMemcpyAsync(host_u,du,sizeof(hipComplex)*m*n*tupe,hipMemcpyDeviceToHost,stream[0]);	
//	hipMemcpy(t,du,sizeof(float)*m*n*tupe,hipMemcpyDeviceToHost);
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);
	hipFree(du);
	hipFree(d_u);


	//itfft_s

	bat = ((m<n)?m:n);
	stride = bat;

	hipComplex* d_s2;
	hipMalloc((void**)&d_s2,sizeof(hipComplex)*ht*bat);
	float* d_s3;
	hipMalloc((void**)&d_s3,sizeof(float)*tupe*bat);
		
	num=bat*ht;
	if(ht*bat<512){
	        threads=num;	
       		blocks=1;
     	}else{
	        threads=512;
	        blocks=((num%512 ==0)?num/512:num/512+1);
	}

    	float2cuComplex<<<blocks,threads,0,stream[1]>>>(ds_extract,ht*bat,d_s2);
	
//	if(hipDeviceSynchronize() != hipSuccess){
//		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
//		return;
//	}
	hipFree(ds_extract);

	hipfftHandle iplan1 =0;
	hipfftSetStream(iplan1,stream[1]);
	
	if (hipfftPlanMany(&iplan1,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}

//	if(hipDeviceSynchronize() != hipSuccess){
//		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
//		return;
//	}

	if(hipfftExecC2R(iplan1,(hipfftComplex*)d_s2,d_s3) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

//	if(hipDeviceSynchronize() != hipSuccess){
//		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
//		return;
//	}
	
       num=bat*tupe;
       if(num<512){
          threads=num;
          blocks=1;
       }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads,0,stream[1]>>>(d_s3,bat*tupe,tupe);

//	hipMemcpy(S,d_s3,sizeof(float)*tupe*bat,hipMemcpyDeviceToHost);
	hipMemcpyAsync(S,d_s3,sizeof(float)*tupe*bat,hipMemcpyDeviceToHost,stream[1]);

	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);

	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftDestroy(iplan1)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_s3);
	hipFree(d_s2);
	
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	
	hipDeviceSynchronize();
}
