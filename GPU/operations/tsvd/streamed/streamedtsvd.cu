#include "svd.h"
#include "based.h"
void streamedtsvd(float* t,const int m,const int n,const int tupe,float* U,float* S,float*  V){
    int bat =m*n;
    hipfftComplex* t_f;

    hipHostAlloc((void**)&t_f,bat*tupe*sizeof(hipfftComplex),hipHostMallocDefault);
    
    //transform t1
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}
/*printf("\n============================\n");
for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f[i].x,t_f[i].y);
}	
printf("\n============================\n");
*/
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
    //process bat
    int bat_num = bat/PLAN1D_SIZE;
    int bat_s = bat%PLAN1D_SIZE;
	hipfftHandle * plan=(hipfftHandle*)malloc(sizeof(hipfftHandle)*PLAN1D_SIZE);
    memset(plan,0,sizeof(hipfftHandle));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&plan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipfftSetStream(plan[i],stream[i]);
    }
    if(bat_num > 0){
    for(int j=0;j<bat_num;j++){

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,t_f+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
     }

    #pragma unroll	
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
            	}
        	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(t_f+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
         }
    }

    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+PLAN1D_SIZE*bat_num*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+tupe*bat_num*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }else{
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe,t_f+i*tupe,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe,d_fftData+i*tupe,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }
	//transform
/*printf("\n============================\n");
for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f[i].x,t_f[i].y);
}	
printf("\n============================\n");
*/
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
/*printf("\n============================\n");
for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f2[i].x,t_f2[i].y);
}	
printf("\n============================\n");
*/	hipMemcpy(d_fftData,t_f2,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);

    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipfftDestroy(plan[i])!=HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }
    }
		
	if(t_f != NULL){
	hipHostFree(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
#if 1//tsvd
	hipsolverHandle_t* handle=(hipsolverHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipsolverHandle_t));
	hipsolverGesvdjInfo_t* params=(hipsolverGesvdjInfo_t*)malloc(tupe*sizeof(hipsolverGesvdjInfo_t));
	int* info = NULL;
	int echo = 1;
	int lda = m;
	int ldu = m;
	int ldv = n;
	int* lwork = (int*)malloc(tupe*sizeof(int));
	hipComplex** work=NULL;

	//malloc u s v

	float* d_s = NULL;
	hipComplex* d_u = NULL;
	hipComplex* d_v = NULL;
	hipMalloc((void**)&d_s,sizeof(float)*tupe*((m<n)?m:n));
	hipMalloc((void**)&d_u,sizeof(hipComplex)*tupe*m*((m<n)?m:n));
	hipMalloc((void**)&d_v,sizeof(hipComplex)*tupe*n*((m<n)?m:n));
	hipMalloc((void**)&info,sizeof(int)*tupe);	
    //set stream
    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipsolverDnCreate(&handle[i]) != HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCreate failed!",__FUNCTION__,__LINE__);
		    return;
    	}
        if(hipsolverSetStream(handle[i],stream[i]) != HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCreate failed!",__FUNCTION__,__LINE__);
		    return;
        }
    
	}
    #pragma unroll
    for(int i=0;i<tupe;i++){	
	if(hipsolverDnCreateGesvdjInfo(&params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:creation svd info error",__FUNCTION__,__LINE__);
		return;
	}
    }	
    int tupe_num=tupe/PLAN1D_SIZE;
    int tupe_s=tupe%PLAN1D_SIZE;
    if(tupe_num > 0){
    for(int j=0;j<tupe_num;j++){
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i+j*PLAN1D_SIZE],
			params[i+j*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }
	for(int i=0;i<tupe_s;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i+tupe_num*PLAN1D_SIZE],
			params[i+tupe_num*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }else{
	for(int i=0;i<tupe_s;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i],
			params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    work=(hipComplex**)malloc(tupe*sizeof(hipComplex*));
    for(int i=0;i<tupe;i++){
	if(hipMalloc((void**)&work[i],sizeof(hipComplex)*lwork[i]) !=hipSuccess){
		fprintf(stdout,"[%s]:[%d] hipMalloc error!",__FUNCTION__,__LINE__);
		return;
    }
    }
   /*for(int i=0;i<tupe;i++){
	if(hipMalloc((void**)&work[i],sizeof(hipComplex)*lwork[i]) !=hipSuccess){
		fprintf(stdout,"[%s]:[%d] hipMalloc error!",__FUNCTION__,__LINE__);
		return;
    }*/
	int step_d = m*n;
	int step_u = m*((m<n)?m:n);
	int step_s = ((m<n)?m:n);
	int step_v = n*((m<n)?m:n);	
    
    if(tupe_num >0){
        for(int j=0;j<tupe_num;j++){
	    for(int i=0;i<PLAN1D_SIZE;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i+j*step_d*PLAN1D_SIZE,
			lda,
			d_s+i*step_s+j*step_s*PLAN1D_SIZE,
			d_u+i*step_u+j*step_u*PLAN1D_SIZE,
			ldu,
			d_v+i*step_v+j*step_v*PLAN1D_SIZE,
			ldv,
			work[i+j*PLAN1D_SIZE],
			lwork[i+j*PLAN1D_SIZE],
			&info[i+j*PLAN1D_SIZE],
			params[i+j*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }   
	    }
        }
	    for(int i=0;i<tupe_s;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i+tupe_num*step_d*PLAN1D_SIZE,
			lda,
			d_s+i*step_s+tupe_num*step_s*PLAN1D_SIZE,
			d_u+i*step_u+tupe_num*step_u*PLAN1D_SIZE,
			ldu,
			d_v+i*step_v+tupe_num*step_v*PLAN1D_SIZE,
			ldv,
			work[i+tupe_num*PLAN1D_SIZE],
			lwork[i+tupe_num*PLAN1D_SIZE],
			&info[i+tupe_num*PLAN1D_SIZE],
			params[i+tupe_num*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }   
	    }
        
    }else{
	    for(int i=0;i<tupe_s;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i,
			lda,
			d_s+i*step_s,
			d_u+i*step_u,
			ldu,
			d_v+i*step_v,
			ldv,
			work[i],
			lwork[i],
			&info[i],
			params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }
	    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipsolverDnDestroy(handle[i])!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }

    for(int i=0;i<tupe;i++){
	if(hipsolverDnDestroyGesvdjInfo(params[i])!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
    }
	
	if(d_fftData != NULL){
	hipFree(d_fftData);
	d_fftData = NULL;
	}
	if(work != NULL){
    for(int i=0;i<tupe;i++){
        hipFree(work[i]);
    }
	hipFree(work);
	work = NULL;
	}
	if(info != NULL){
	hipFree(info);
	info = NULL;
	}
#endif
	//ifft
	//transform
//	hipComplex* h_u = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_u);
    hipComplex* h_u;
    hipHostAlloc((void**)&h_u,tupe*step_u*sizeof(hipComplex),hipHostMallocDefault);
    hipComplex* h_u2 = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_u);

//	hipComplex* h_v = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_v);
    hipComplex* h_v;
    hipHostAlloc((void**)&h_v,tupe*step_v*sizeof(hipComplex),hipHostMallocDefault);
    hipComplex* h_v2 = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_v);

//	hipComplex* h_s = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_s);
    hipComplex* h_s;
    hipHostAlloc((void**)&h_s,tupe*step_s*sizeof(hipComplex),hipHostMallocDefault);
    float* h_s2 = (float*)malloc(sizeof(float)*tupe*step_s);

	hipComplex* d_s2;
	hipMalloc((void**)&d_s2,sizeof(hipComplex)*tupe*step_s);

	hipMemcpy(h_u2,d_u,sizeof(hipComplex)*tupe*step_u,hipMemcpyDeviceToHost);
	hipMemcpy(h_v2,d_v,sizeof(hipComplex)*tupe*step_v,hipMemcpyDeviceToHost);
	hipMemcpy(h_s2,d_s,sizeof(float)*tupe*step_s,hipMemcpyDeviceToHost);
	
/*printf("\n============================\n");
for(int i=0;i<step_s*tupe;i++){
    printf("[%f ]",h_s2[i]);
}	
printf("\n============================\n");
*/	//transform_u
	for(int i=0;i<step_u;i++){
	  for(int j=0;j<tupe;j++){
		h_u[i*tupe+j]=h_u2[j*step_u+i];
		}
	}

	//transform_v
	for(int i=0;i<step_v;i++){
	  for(int j=0;j<tupe;j++){
		h_v[i*tupe+j]=h_v2[j*step_v+i];
		}
	}
		
	//transform_s
	for(int i=0;i<step_s;i++){
	  for(int j=0;j<tupe;j++){
		h_s[i*tupe+j].x=h_s2[j*step_s+i];
		h_s[i*tupe+j].y=0;
		}
	}
	
/*	for(int i=0;i<tupe*step_s;i++){
		printf("%f ",h_s2[i]);
	}
	printf("\n");
*/
	
	if(h_u2 != NULL){
	free(h_u2);
	h_u2 = NULL;
	}
	if(h_v2 != NULL){
	free(h_v2);
	h_v2 = NULL;
	}
	if(h_s2 != NULL){
	free(h_s2);
	h_s2= NULL;
	}
	if(d_s != NULL){
	hipFree(d_s);
	d_s = NULL;
	}

	hipfftHandle* iplan =(hipfftHandle*)malloc(PLAN1D_SIZE*sizeof(hipfftHandle));
    for(int i=0;i<PLAN1D_SIZE;i++){

	if(hipfftPlan1d(&iplan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		    return;	
	}

    if(hipfftSetStream(iplan[i],stream[i]) != HIPFFT_SUCCESS){            
		    fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		    return;	
        }
	}

//	hipMemcpy(d_s2,h_s,sizeof(hipComplex)*tupe*step_s,hipMemcpyHostToDevice);
//	hipMemcpy(d_v,h_v,sizeof(hipComplex)*tupe*step_v,hipMemcpyHostToDevice);
	//ifft_u
    int step_u_num=step_u/PLAN1D_SIZE;
    int step_u_s=step_u%PLAN1D_SIZE;
    if(step_u_num > 0){
    for(int j=0;j<step_u_num;j++){
    for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipMemcpyAsync(d_u+i*tupe+j*tupe*PLAN1D_SIZE,h_u+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
                }
	}
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_u+i*tupe+j*tupe*PLAN1D_SIZE,d_u+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(h_u+i*tupe+j*tupe*PLAN1D_SIZE,d_u+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
        }
    }
    }
    for(int i=0;i<step_u_s;i++){
    if(hipMemcpyAsync(d_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,h_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
                }
	}
    for(int i=0;i<step_u_s;i++){
	if(hipfftExecC2C(iplan[i],d_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,d_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    for(int i=0;i<step_u_s;i++){
	if(hipMemcpyAsync(h_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,d_u+i*tupe+step_u_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
        }
    }
    }else{
    for(int i=0;i<step_u_s;i++){
    if(hipMemcpyAsync(d_u+i*tupe,h_u+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
                }
	}
    for(int i=0;i<step_u_s;i++){
	if(hipfftExecC2C(iplan[i],d_u+i*tupe,d_u+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    for(int i=0;i<step_u_s;i++){
	if(hipMemcpyAsync(h_u+i*tupe,d_u+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
        }
    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
	//ifft_v
    int step_v_num=step_v/PLAN1D_SIZE;
    int step_v_s=step_v%PLAN1D_SIZE;
    if(step_v_num > 0){
    for(int j=0;j<step_v_num;j++){
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(d_v+i*tupe+j*tupe*PLAN1D_SIZE,h_v+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }       
	    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_v+i*tupe+j*tupe*PLAN1D_SIZE,d_v+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]CUFFT ERROR: cufftExecc2Cfailed!",__FUNCTION__,__LINE__);
		return;
	        }
	    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(h_v+i*tupe+j*tupe*PLAN1D_SIZE,d_v+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
        }
    }
    for(int i=0;i<step_v_s;i++){
	if(hipMemcpyAsync(d_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,h_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }       
	    }
    for(int i=0;i<step_v_s;i++){
	if(hipfftExecC2C(iplan[i],d_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,d_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]CUFFT ERROR: cufftExecc2Cfailed!",__FUNCTION__,__LINE__);
		return;
	        }
	    }
    for(int i=0;i<step_v_s;i++){
	if(hipMemcpyAsync(h_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,d_v+i*tupe+step_v_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
        }
    }else{
    for(int i=0;i<step_v_s;i++){
	if(hipMemcpyAsync(d_v+i*tupe,h_v+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }       
	    }
    for(int i=0;i<step_v_s;i++){
	if(hipfftExecC2C(iplan[i],d_v+i*tupe,d_v+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]CUFFT ERROR: cufftExecc2Cfailed!",__FUNCTION__,__LINE__);
		return;
	        }
	    }
    for(int i=0;i<step_v_s;i++){
	if(hipMemcpyAsync(h_v+i*tupe,d_v+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
        }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
	
	//ifft_s
    int step_s_num=step_s/PLAN1D_SIZE;
    int step_s_s=step_s%PLAN1D_SIZE;
    if(step_s_num > 0){
    for(int j=0;j<step_s_num;j++){
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(d_s2+i*tupe+j*tupe*PLAN1D_SIZE,h_s+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
	    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_s2+i*tupe+j*tupe*PLAN1D_SIZE,d_s2+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	
	        }
	    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(h_s+i*tupe+j*tupe*PLAN1D_SIZE,d_s2+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
	    }
       }
    for(int i=0;i<step_s_s;i++){
	if(hipMemcpyAsync(d_s2+i*tupe+step_s_num*tupe*PLAN1D_SIZE,h_s+i*tupe+step_s_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
	    }
    for(int i=0;i<step_s_s;i++){
	if(hipfftExecC2C(iplan[i],d_s2+i*tupe+step_s_num*tupe*PLAN1D_SIZE,d_s2+i*tupe+step_s_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	
	        }
	    }
    for(int i=0;i<step_s_s;i++){
	if(hipMemcpyAsync(h_s+i*tupe+step_s_num*tupe*PLAN1D_SIZE,d_s2+i*tupe+step_s_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
    }
    }else{
    for(int i=0;i<step_s_s;i++){
	if(hipMemcpyAsync(d_s2+i*tupe,h_s+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]) !=hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
	    }
    for(int i=0;i<step_s_s;i++){
	if(hipfftExecC2C(iplan[i],d_s2+i*tupe,d_s2+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	
	        }
	    }
    for(int i=0;i<step_s_s;i++){
	if(hipMemcpyAsync(h_s+i*tupe,d_s2+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
		        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
	    	    return;
            }
    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        if(hipStreamSynchronize(stream[i]) !=hipSuccess){
        	fprintf(stdout,"[%s]:[%d] hipStreamSynchronize error!",__FUNCTION__,__LINE__);
        	return;	
            }
        }
	//transform_u
	for(int i=0;i<step_u;i++){
	  for(int j=0;j<tupe;j++){
		U[j*step_u+i]=h_u[i*tupe+j].x/tupe;
	//	U[j*step_u+i].y=h_u[i*tupe+j].y/tupe;
		}
	}

	//transform_v
	for(int i=0;i<step_v;i++){
	  for(int j=0;j<tupe;j++){
		V[j*step_v+i]=h_v[i*tupe+j].x/tupe;
	//	V[j*step_v+i].y=h_v[i*tupe+j].y/tupe;
		}
	}	

	//transform_s
	for(int i=0;i<step_s;i++){
	  for(int j=0;j<tupe;j++){
		S[j*step_s+i]=h_s[i*tupe+j].x/tupe;
	//	S[j*step_s+j].y=h_s[i*tupe+j].y/tupe;
		}
	}

	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftDestroy(iplan[i])!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
    if(hipStreamDestroy(stream[i]) != hipSuccess){
        	fprintf(stdout,"[%s]:[%d] destory stream error!",__FUNCTION__,__LINE__);
        	return;	
    }
    }

	if(d_u != NULL){	
	hipFree(d_u);
	d_u =NULL;
	}
	if(d_v != NULL){
	hipFree(d_v);
	d_v = NULL;
	}
	if(d_s2 != NULL){
	hipFree(d_s2);
	d_s2 = NULL;
	}
	if(h_u !=NULL){
	hipHostFree(h_u);
	h_u = NULL;
	}
	if(h_v != NULL){
	hipHostFree(h_v);
	h_v = NULL;
	}
	if(h_s != NULL){
	hipHostFree(h_s);
	h_s = NULL;
    }
}
