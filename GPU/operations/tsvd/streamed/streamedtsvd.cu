#include "svd.h"
#include "based.h"
void streamedtsvd(float* t,const int m,const int n,const int tupe,float* U,float* S,float*  V){
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
/*	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));
*/
	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_t);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

#if 1//tsvd
	hipsolverHandle_t* handle=(hipsolverHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipsolverHandle_t));
	hipsolverGesvdjInfo_t* params=(hipsolverGesvdjInfo_t*)malloc(ht*sizeof(hipsolverGesvdjInfo_t));
	int* info = NULL;
	int echo = 1;
	int lda = m;
	int ldu = m;
	int ldv = n;
	int* lwork = (int*)malloc(ht*sizeof(int));
	hipComplex** work=NULL;

	//malloc u s v

	float* d_s = NULL;
	hipComplex* d_u = NULL;
	hipComplex* d_v = NULL;
	hipMalloc((void**)&d_s,sizeof(float)*ht*((m<n)?m:n));
	hipMalloc((void**)&d_u,sizeof(hipComplex)*ht*m*((m<n)?m:n));
	hipMalloc((void**)&d_v,sizeof(hipComplex)*ht*n*((m<n)?m:n));
	hipMalloc((void**)&info,sizeof(int)*ht);	
    //set stream
    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipsolverDnCreate(&handle[i]) != HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCreate failed!",__FUNCTION__,__LINE__);
		    return;
    	}
        if(hipsolverSetStream(handle[i],stream[i]) != HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCreate failed!",__FUNCTION__,__LINE__);
		    return;
        }
    
	}
    #pragma unroll
    for(int i=0;i<ht;i++){	
	if(hipsolverDnCreateGesvdjInfo(&params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:creation svd info error",__FUNCTION__,__LINE__);
		return;
	}
    }	
    int tupe_num=ht/PLAN1D_SIZE;
    int tupe_s=ht%PLAN1D_SIZE;
    if(tupe_num > 0){
    for(int j=0;j<tupe_num;j++){
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i+j*PLAN1D_SIZE],
			params[i+j*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }
	for(int i=0;i<tupe_s;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i+tupe_num*PLAN1D_SIZE],
			params[i+tupe_num*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }else{
	for(int i=0;i<tupe_s;i++){
	if(hipsolverDnCgesvdj_bufferSize(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork[i],
			params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	        }
        }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    work=(hipComplex**)malloc(ht*sizeof(hipComplex*));
    for(int i=0;i<ht;i++){
	if(hipMalloc((void**)&work[i],sizeof(hipComplex)*lwork[i]) !=hipSuccess){
		fprintf(stdout,"[%s]:[%d] hipMalloc error!",__FUNCTION__,__LINE__);
		return;
    }
    }
   /*for(int i=0;i<tupe;i++){
	if(hipMalloc((void**)&work[i],sizeof(hipComplex)*lwork[i]) !=hipSuccess){
		fprintf(stdout,"[%s]:[%d] hipMalloc error!",__FUNCTION__,__LINE__);
		return;
    }*/
	int step_d = m*n;
	int step_u = m*((m<n)?m:n);
	int step_s = ((m<n)?m:n);
	int step_v = n*((m<n)?m:n);	
    
    if(tupe_num >0){
        for(int j=0;j<tupe_num;j++){
	    for(int i=0;i<PLAN1D_SIZE;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i+j*step_d*PLAN1D_SIZE,
			lda,
			d_s+i*step_s+j*step_s*PLAN1D_SIZE,
			d_u+i*step_u+j*step_u*PLAN1D_SIZE,
			ldu,
			d_v+i*step_v+j*step_v*PLAN1D_SIZE,
			ldv,
			work[i+j*PLAN1D_SIZE],
			lwork[i+j*PLAN1D_SIZE],
			&info[i+j*PLAN1D_SIZE],
			params[i+j*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }   
	    }
        }
	    for(int i=0;i<tupe_s;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i+tupe_num*step_d*PLAN1D_SIZE,
			lda,
			d_s+i*step_s+tupe_num*step_s*PLAN1D_SIZE,
			d_u+i*step_u+tupe_num*step_u*PLAN1D_SIZE,
			ldu,
			d_v+i*step_v+tupe_num*step_v*PLAN1D_SIZE,
			ldv,
			work[i+tupe_num*PLAN1D_SIZE],
			lwork[i+tupe_num*PLAN1D_SIZE],
			&info[i+tupe_num*PLAN1D_SIZE],
			params[i+tupe_num*PLAN1D_SIZE]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }   
	    }
        
    }else{
	    for(int i=0;i<tupe_s;i++){
	    if(hipsolverDnCgesvdj(
			handle[i],
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i,
			lda,
			d_s+i*step_s,
			d_u+i*step_u,
			ldu,
			d_v+i*step_v,
			ldv,
			work[i],
			lwork[i],
			&info[i],
			params[i]) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		    }
	    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipsolverDnDestroy(handle[i])!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }

    for(int i=0;i<ht;i++){
	if(hipsolverDnDestroyGesvdjInfo(params[i])!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
    }
	
	if(d_fftData != NULL){
	hipFree(d_fftData);
	d_fftData = NULL;
	}
	if(work != NULL){
    for(int i=0;i<tupe;i++){
        hipFree(work[i]);
    }
	hipFree(work);
	work = NULL;
	}
	if(info != NULL){
	hipFree(info);
	info = NULL;
	}
#endif

	for(int i=0;i<PLAN1D_SIZE;i++){
    if(hipStreamDestroy(stream[i]) != hipSuccess){
        	fprintf(stdout,"[%s]:[%d] destory stream error!",__FUNCTION__,__LINE__);
        	return;	
    }
    }
    //ifft_u
	int threads=0;
	int blocks=0;
	
	hipfftHandle iplan =0;
	in[0] = ht;
	on[0] = tupe;
	bat = m*((m<n)?m:n);
	stride = bat;
	float* du;
	hipMalloc((void**)&du,sizeof(float)*bat*tupe);
	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	//printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecC2R(iplan,(hipfftComplex*)d_u,du)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	int num=0;

	num=bat*tupe;
        if(num<512){
          threads=num;
          blocks=1;
        }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads>>>(du,num,tupe);
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	hipMemcpy(U,du,sizeof(float)*bat*tupe,hipMemcpyDeviceToHost);
	
	hipFree(du);
	hipFree(d_u);

	//ifft_v
	
	in[0] = ht;
	on[0] = tupe;
	bat = n*((m<n)?m:n);
	stride = bat;
	
	float* dv;
	hipMalloc((void**)&dv,sizeof(float)*bat*tupe);
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	//printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecC2R(iplan,(hipfftComplex*)d_v,dv)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	num=bat*tupe;
        if(num<512){
          threads=num;
          blocks=1;
        }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads>>>(dv,num,tupe);
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	hipMemcpy(V,dv,sizeof(float)*bat*tupe,hipMemcpyDeviceToHost);
	
	hipFree(dv);
	hipFree(d_v);
	//ifft_s

	bat = ((m<n)?m:n);
	stride = bat;

	hipComplex* d_s2;
	hipMalloc((void**)&d_s2,sizeof(hipComplex)*ht*bat);
	float* d_s3;
	hipMalloc((void**)&d_s3,sizeof(float)*tupe*bat);
		
	num=bat*ht;
	if(ht*bat<512){
	        threads=num;	
       		blocks=1;
     	}else{
	        threads=512;
	        blocks=((num%512 ==0)?num/512:num/512+1);
	}

    	float2cuComplex<<<blocks,threads>>>(d_s,ht*bat,d_s2);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	hipFree(d_s);

	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipfftExecC2R(iplan,(hipfftComplex*)d_s2,d_s3) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
       num=bat*tupe;
       if(num<512){
          threads=num;
          blocks=1;
       }else{
	  threads=512;
	  blocks=((num%512 ==0)?num/512:num/512+1);
	}
         fftResultProcess<<<blocks,threads>>>(d_s3,bat*tupe,tupe);
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(S,d_s3,sizeof(float)*tupe*bat,hipMemcpyDeviceToHost);


	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	hipFree(d_s3);
	hipFree(d_s2);
	
}
