#include "svd.h"
#include "based.h"
void basedtsvd(float* t,const int m,const int n,const int tupe,float* U,float* S,float*  V){
	int bat = m*n;
	hipfftComplex* t_f = (hipfftComplex*)malloc(bat*tupe*sizeof(hipfftComplex));
	//transform
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
	hipMemcpy(d_fftData,t_f,bat*tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	hipfftHandle plan;
	if(hipfftPlan1d(&plan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(plan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}

	//transform
	hipMemcpy(t_f,d_fftData,sizeof(hipfftComplex)*bat*tupe,hipMemcpyDeviceToHost);
	hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
	
/*printf("\n============================\n");
for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f2[i].x,t_f2[i].y);
}	
printf("\n============================\n");
*/	hipMemcpy(d_fftData,t_f2,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);
	
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
		
	if(t_f != NULL){
	free(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
	//tsvd
	hipsolverHandle_t handle;
	hipsolverGesvdjInfo_t params;
	int* info = NULL;
	int echo = 1;
	int lda = m;
	int ldu = m;
	int ldv = n;
	int lwork = 0;
	hipComplex* work=NULL;

	//malloc u s v

	float* d_s = NULL;
	hipComplex* d_u = NULL;
	hipComplex* d_v = NULL;
	hipMalloc((void**)&d_s,sizeof(float)*tupe*((m<n)?m:n));
	hipMalloc((void**)&d_u,sizeof(hipComplex)*tupe*m*((m<n)?m:n));
	hipMalloc((void**)&d_v,sizeof(hipComplex)*tupe*n*((m<n)?m:n));
	hipMalloc((void**)&info,sizeof(int));	
	
	if(hipsolverDnCreate(&handle) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnCreate failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(hipsolverDnCreateGesvdjInfo(&params) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:creation svd info srror",__FUNCTION__,__LINE__);
		return;
	}	
	
	if(hipsolverDnCgesvdj_bufferSize(
			handle,
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData,
			m,
			d_s,
			d_u,
			ldu,
			d_v,
			ldv,
			&lwork,
			params) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR: create buffersize failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMalloc((void**)&work,sizeof(hipComplex)*lwork);

	int step_d = m*n;
	int step_u = m*((m<n)?m:n);
	int step_s = ((m<n)?m:n);
	int step_v = n*((m<n)?m:n);	
	
	for(int i=0;i<tupe;i++){
	  if(hipsolverDnCgesvdj(
			handle,
			HIPSOLVER_EIG_MODE_VECTOR,
			echo,
			m,
			n,
			d_fftData+step_d*i,
			lda,
			d_s+i*step_s,
			d_u+i*step_u,
			ldu,
			d_v+i*step_v,
			ldv,
			work,
			lwork,
			info,
			params) != HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUSOLVER ERROR:hipsolverDnCgesvdj failed!",__FUNCTION__,__LINE__);
		return;
		}
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipsolverDnDestroy(handle)!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipsolverDnDestroyGesvdjInfo(params)!=HIPSOLVER_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipsolverDnDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(d_fftData != NULL){
	hipFree(d_fftData);
	d_fftData = NULL;
	}
	if(work != NULL){
	hipFree(work);
	work = NULL;
	}
	if(info != NULL){
	hipFree(info);
	info = NULL;
	}

	//ifft
	
	//transform
	hipComplex* h_u = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_u);
	hipComplex* h_u2 = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_u);
	hipComplex* h_v = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_v);
	hipComplex* h_v2 = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_v);
	hipComplex* h_s = (hipComplex*)malloc(sizeof(hipComplex)*tupe*step_s);
	float* h_s2 = (float*)malloc(sizeof(float)*tupe*step_s);
	hipComplex* d_s2;
	hipMalloc((void**)&d_s2,sizeof(hipComplex)*tupe*step_s);

	hipMemcpy(h_u2,d_u,sizeof(hipComplex)*tupe*step_u,hipMemcpyDeviceToHost);
	hipMemcpy(h_v2,d_v,sizeof(hipComplex)*tupe*step_v,hipMemcpyDeviceToHost);
	hipMemcpy(h_s2,d_s,sizeof(float)*tupe*step_s,hipMemcpyDeviceToHost);
/*printf("\n============================\n");
for(int i=0;i<tupe*step_s;i++){
    printf("[%f ]",h_s2[i]);
}	
printf("\n============================\n");
*/	
	//transform_u
	for(int i=0;i<step_u;i++){
	  for(int j=0;j<tupe;j++){
		h_u[i*tupe+j]=h_u2[j*step_u+i];
		}
	}

	//transform_v
	for(int i=0;i<step_v;i++){
	  for(int j=0;j<tupe;j++){
		h_v[i*tupe+j]=h_v2[j*step_v+i];
		}
	}
		
	//transform_s
	for(int i=0;i<step_s;i++){
	  for(int j=0;j<tupe;j++){
		h_s[i*tupe+j].x=h_s2[j*step_s+i];
		h_s[i*tupe+j].y=0;
		}
	}
	
/*	for(int i=0;i<tupe*step_s;i++){
		printf("%f ",h_s2[i]);
	}
	printf("\n");
*/	hipMemcpy(d_u,h_u,sizeof(hipComplex)*tupe*step_u,hipMemcpyHostToDevice);
	hipMemcpy(d_s2,h_s,sizeof(hipComplex)*tupe*step_s,hipMemcpyHostToDevice);
	hipMemcpy(d_v,h_v,sizeof(hipComplex)*tupe*step_v,hipMemcpyHostToDevice);
	
	if(h_u2 != NULL){
	free(h_u2);
	h_u2 = NULL;
	}
	if(h_v2 != NULL){
	free(h_v2);
	h_v2 = NULL;
	}
	if(h_s2 != NULL){
	free(h_s2);
	h_s2= NULL;
	}
	if(d_s != NULL){
	hipFree(d_s);
	d_s = NULL;
	}

	hipfftHandle iplan;

	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//ifft_u
	for(int i=0;i<step_u;i++){
	if(hipfftExecC2C(iplan,d_u+i*tupe,d_u+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//ifft_v
	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	for(int i=0;i<step_v;i++){
	if(hipfftExecC2C(iplan,d_v+i*tupe,d_v+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]CUFFT ERROR: cufftExecc2Cfailed!",__FUNCTION__,__LINE__);
		return;
	}
	}

	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//ifft_s
	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	for(int i=0;i<step_s;i++){
	if(hipfftExecC2C(iplan,d_s2+i*tupe,d_s2+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	
	}
	}
	
	//transform
		
	hipMemcpy(h_u,d_u,sizeof(hipComplex)*tupe*step_u,hipMemcpyDeviceToHost);
	hipMemcpy(h_v,d_v,sizeof(hipComplex)*tupe*step_v,hipMemcpyDeviceToHost);
	hipMemcpy(h_s,d_s2,sizeof(hipComplex)*tupe*step_s,hipMemcpyDeviceToHost);

	//transform_u
	for(int i=0;i<step_u;i++){
	  for(int j=0;j<tupe;j++){
		U[j*step_u+i]=h_u[i*tupe+j].x/tupe;
	//	U[j*step_u+i].y=h_u[i*tupe+j].y/tupe;
		}
	}

	//transform_v
	for(int i=0;i<step_v;i++){
	  for(int j=0;j<tupe;j++){
		V[j*step_v+i]=h_v[i*tupe+j].x/tupe;
	//	V[j*step_v+i].y=h_v[i*tupe+j].y/tupe;
		}
	}	

	//transform_s
	for(int i=0;i<step_s;i++){
	  for(int j=0;j<tupe;j++){
		S[j*step_s+i]=h_s[i*tupe+j].x/tupe;
	//	S[j*step_s+j].y=h_s[i*tupe+j].y/tupe;
		}
	}
	
	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(d_u != NULL){	
	hipFree(d_u);
	d_u =NULL;
	}
	if(d_v != NULL){
	hipFree(d_v);
	d_v = NULL;
	}
	if(d_s2 != NULL){
	hipFree(d_s2);
	d_s2 = NULL;
	}
	if(h_u !=NULL){
	free(h_u);
	h_u = NULL;
	}
	if(h_v != NULL){
	free(h_v);
	h_v = NULL;
	}
	if(h_s != NULL){
	free(h_s);
	h_s = NULL;
	}
}
