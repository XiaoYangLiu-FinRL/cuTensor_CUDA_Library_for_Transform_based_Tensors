#include "qr.h"
void  basedtqr(float *A,const int m,const int n,const int tupe, hipComplex* Tau)
{	
	int bat = m*n;
	hipfftComplex* t_f = (hipfftComplex*)malloc(bat*tupe*sizeof(hipfftComplex));
	//transform
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=A[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
	hipMemcpy(d_fftData,t_f,bat*tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	hipfftHandle plan;
	if(hipfftPlan1d(&plan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(plan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
    }

	//transform
	hipMemcpy(t_f,d_fftData,sizeof(hipfftComplex)*bat*tupe,hipMemcpyDeviceToHost);
	hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
	
	hipMemcpy(d_fftData,t_f2,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);
	
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
		
	if(t_f != NULL){
	free(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
   
	// qr
	hipsolverHandle_t cusolverH;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

	hipError_t cudaStat2 = hipSuccess ;
	hipError_t cudaStat3 = hipSuccess ;
       
     
	hipComplex *d_tau = NULL;
	hipComplex *d_work=NULL;
        int *devInfo=NULL;
	int lda = m;
        int lwork = 0;
        int info_gpu = 0;
	int strid_A=m*n;
	int tau=imin(m,n);
        cusolver_status = hipsolverDnCreate(&cusolverH);
    
        cudaStat2 = hipMalloc (( void **)& d_tau , sizeof ( hipComplex ) * tupe * tau);
        cudaStat3 = hipMalloc (( void **)& devInfo , sizeof ( int ));
	if(cudaStat2 != hipSuccess|| cudaStat3 != hipSuccess){
	}
       
        cusolver_status = hipsolverDnCgeqrf_bufferSize(cusolverH, m, n, d_fftData, lda, &lwork);
	hipDeviceSynchronize();

        hipMalloc (( void **)& d_work , sizeof ( hipComplex )* lwork );
	 
	for(int i=0;i<tupe;i++){
        cusolver_status = hipsolverDnCgeqrf(cusolverH, m, n, d_fftData+i*strid_A, lda, d_tau+i*tau, d_work, lwork, devInfo);
        hipDeviceSynchronize();
	if(cusolver_status !=HIPSOLVER_STATUS_SUCCESS){
	fprintf(stderr,"[%s]:[%d]ERROR!",__FUNCTION__,__LINE__);
	}

	hipMemcpy(Tau,d_tau,sizeof(hipComplex)* tupe * tau,hipMemcpyDeviceToHost);
	
        hipMemcpy (&info_gpu , devInfo , sizeof ( int ) , hipMemcpyDeviceToHost);
        printf("after geqrf:info_gpu = %d\n", info_gpu);
	}
	//Tifft

	//transform
	hipComplex* h_fftData = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);
	hipComplex* h_fftData1 = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);
	hipMemcpy(h_fftData, d_fftData, sizeof(hipComplex) *bat *tupe,hipMemcpyDeviceToHost);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		h_fftData1[i*tupe+j]=h_fftData[j*bat+i];
		}
	} 
	hipMemcpy(d_fftData, h_fftData1, sizeof(hipComplex)*tupe*bat,hipMemcpyHostToDevice);
	
	if(h_fftData != NULL){
	free(h_fftData);
	h_fftData = NULL;
	}

	if(h_fftData1 != NULL){
	free(h_fftData1);
	h_fftData1 = NULL;
	}

	hipfftHandle iplan;

	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(iplan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
    }
    
	hipDeviceSynchronize();

	hipComplex* h_A = (hipComplex*)malloc(sizeof(hipComplex)*tupe*bat);
	hipMemcpy(h_A,d_fftData,tupe*bat*sizeof(hipComplex),hipMemcpyDeviceToHost);
	
	hipFree(d_fftData);
 	
	//transform
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		A[j*bat+i]=h_A[i*tupe+j].x/tupe;
		}
	}
	
	hipfftDestroy(iplan);
        hipFree(d_tau);
	free(h_A);
        hipFree(devInfo);
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
}
