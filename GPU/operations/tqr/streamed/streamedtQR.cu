#include "qr.h"
#include "based.h"
void  streamedtqr(float *A,const int m,const int n,const int tupe, hipComplex* Tau)
{	
	
    int bat =m*n;
    hipfftComplex* t_f;

    hipHostAlloc((void**)&t_f,bat*tupe*sizeof(hipfftComplex),hipHostMallocDefault);
    
    //transform t1
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=A[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
    //process bat
    int bat_num = bat/PLAN1D_SIZE;
    int bat_s = bat%PLAN1D_SIZE;
	hipfftHandle * plan=(hipfftHandle*)malloc(sizeof(hipfftHandle)*PLAN1D_SIZE);
    memset(plan,0,sizeof(hipfftHandle));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&plan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipfftSetStream(plan[i],stream[i]);
    }
    if(bat_num > 0){
    for(int j=0;j<bat_num;j++){

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,t_f+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]) != hipSuccess){
        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
        return;
    }
     }

    #pragma unroll	
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
            	}
        	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipMemcpyAsync(t_f+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]) != hipSuccess){
        fprintf(stdout,"[%s]:[%d] hipMemcpyAsync error!",__FUNCTION__,__LINE__);
        return;
    }
         }
    }

    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+PLAN1D_SIZE*bat_num*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+tupe*bat_num*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }else{
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe,t_f+i*tupe,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe,d_fftData+i*tupe,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
	//transform
    hipfftComplex* t_f2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f2[j*bat+i]=t_f[i*tupe+j];
		}
	}
    /*printf("\n============================\n");
    for(int i=0;i<bat*tupe;i++){
    printf("[%f %f]",t_f2[i].x,t_f2[i].y);
    }	
    printf("\n============================\n");
    */	
    
    hipMemcpy(d_fftData,t_f2,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);

    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipfftDestroy(plan[i])!=HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }
    }
		
	if(t_f != NULL){
	hipHostFree(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
   
	// qr
	hipsolverHandle_t* cusolverH=(hipsolverHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipsolverHandle_t));
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

       
     
	hipComplex *d_tau = NULL;
	hipComplex **d_work=NULL;
    int** devInfo=(int**)malloc(PLAN1D_SIZE*sizeof(int*));
	int lda = m;
    int lwork = 0;
	int strid_A=m*n;
	int tau=imin(m,n);
   
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    cusolver_status = hipsolverDnCreate(&cusolverH[i]);
    hipsolverSetStream(cusolverH[i],stream[i]);
    }

    hipMalloc (( void **)& d_tau , sizeof ( hipComplex ) * tupe * tau);
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    hipMalloc (( void **)& devInfo[i] , sizeof ( int ));
	}
       
    hipsolverDnCgeqrf_bufferSize(cusolverH[0], m, n, d_fftData, lda, &lwork);

    d_work=(hipComplex**)malloc(tupe*sizeof(hipComplex*));
    #pragma unroll
    for(int i=0;i<tupe;i++){
    hipMalloc (( void **)& d_work[i] , sizeof ( hipComplex )* lwork );
	}
    int tupe_num= tupe/PLAN1D_SIZE;
    int tupe_s= tupe%PLAN1D_SIZE;
    if(tupe_num > 0){
    #pragma unroll
    for(int j=0;j<tupe_num;j++){
    #pragma unroll
	for(int i=0;i<PLAN1D_SIZE;i++){
       if( hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A+j*strid_A*PLAN1D_SIZE, lda, d_tau+i*tau+j*tau*PLAN1D_SIZE, d_work[i+j*PLAN1D_SIZE], lwork, devInfo[i]) !=HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCgeqrf error!",__FUNCTION__,__LINE__);
		    return;
       }
        }
     }
    #pragma unroll
	for(int i=0;i<tupe_s;i++){
        cusolver_status = hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A+tupe_num*strid_A*PLAN1D_SIZE, lda, d_tau+i*tau+tupe_num*tau*PLAN1D_SIZE, d_work[i+tupe_num*PLAN1D_SIZE], lwork, devInfo[i]);
	if(cusolver_status !=HIPSOLVER_STATUS_SUCCESS){
	fprintf(stderr,"[%s]:[%d]ERROR!",__FUNCTION__,__LINE__);
		    return;
	        }
        }
    }else{
    #pragma unroll
	for(int i=0;i<tupe_s;i++){
        cusolver_status = hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A, lda, d_tau+i*tau, d_work[i], lwork, devInfo[i]);
	if(cusolver_status !=HIPSOLVER_STATUS_SUCCESS){
	fprintf(stderr,"[%s]:[%d]ERROR!",__FUNCTION__,__LINE__);
		    return;
	        }
    }
    }
	hipMemcpy(Tau,d_tau,sizeof(hipComplex)* tupe * tau,hipMemcpyDeviceToHost);

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    hipsolverDnDestroy(cusolverH[i]);
        }
	//Tifft

	//transform
	hipComplex* h_fftData = (hipComplex*)malloc(sizeof(hipComplex) * bat * tupe);
    hipComplex* h_A;
    hipHostAlloc((void**)&h_A,tupe*bat*sizeof(hipComplex),hipHostMallocDefault);

    hipMemcpy(h_fftData, d_fftData, sizeof(hipComplex) *bat *tupe,hipMemcpyDeviceToHost);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		h_A[i*tupe+j]=h_fftData[j*bat+i];
		}
	} 
	
	if(h_fftData != NULL){
	free(h_fftData);
	h_fftData = NULL;
	}
    
	hipfftHandle* iplan=(hipfftHandle*)malloc(PLAN1D_SIZE*sizeof(hipfftHandle));
    
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&iplan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
    if(hipfftSetStream(iplan[i],stream[i]) !=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] cufft set stream error!",__FUNCTION__,__LINE__);
		return;	
    }
    }
	
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
    bat_num=bat/PLAN1D_SIZE;
    bat_s=bat%PLAN1D_SIZE;
    if(bat_num > 0){
    #pragma unroll
    for(int j=0;j<bat_num;j++){
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE, h_A+i*tupe+j*tupe*PLAN1D_SIZE, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(h_A+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
        }
    }
    #pragma unroll
    for(int i=0;i<bat_s;i++){
    hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE, h_A+i*tupe+bat_num*tupe*PLAN1D_SIZE, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(h_A+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
    }
    
    }else{
    #pragma unroll
    for(int i=0;i<bat_s;i++){
    hipMemcpyAsync(d_fftData+i*tupe, h_A+i*tupe, sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(iplan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
    }
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(h_A+i*tupe,d_fftData+i*tupe,tupe*sizeof(hipComplex),hipMemcpyDeviceToHost,stream[i]);
    }
    }
	
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
 	
	//transform
	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		A[j*bat+i]=h_A[i*tupe+j].x/tupe;
		}
	}

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){	
	    if(hipfftDestroy(iplan[i])!=HIPFFT_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		    return;
	    }
    }
	
	hipFree(d_fftData);
    hipFree(d_tau);

	hipHostFree(h_A);
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){	
    hipFree(devInfo[i]);
    }
    free(devInfo);
    #pragma unroll
    for(int i=0;i<tupe;i++){	
    hipFree(d_work[i]);
    }
    free(d_work);
    hipDeviceReset();
}
