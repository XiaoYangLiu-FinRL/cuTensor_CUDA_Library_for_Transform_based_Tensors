#include "qr.h"
#include "based.h"
void  streamedtqr(float *A,const int m,const int n,const int tupe, hipComplex* Tau)
{	
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,A,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	if(d_t !=NULL){
	hipFree(d_t);
        d_t=NULL;   
        }
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}
	
    //set stream for t
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	// qr
	hipsolverHandle_t* cusolverH=(hipsolverHandle_t*)malloc(PLAN1D_SIZE*sizeof(hipsolverHandle_t));
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

       
     
	hipComplex *d_tau = NULL;
	hipComplex **d_work=NULL;
    int** devInfo=(int**)malloc(PLAN1D_SIZE*sizeof(int*));
	int lda = m;
    int lwork = 0;
	int strid_A=m*n;
	int tau=imin(m,n);
   
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    cusolver_status = hipsolverDnCreate(&cusolverH[i]);
    hipsolverSetStream(cusolverH[i],stream[i]);
    }

    hipMalloc (( void **)& d_tau , sizeof ( hipComplex ) * ht * tau);
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    hipMalloc (( void **)& devInfo[i] , sizeof ( int ));
	}
       
    hipsolverDnCgeqrf_bufferSize(cusolverH[0], m, n, d_fftData, lda, &lwork);

    d_work=(hipComplex**)malloc(ht*sizeof(hipComplex*));
    #pragma unroll
    for(int i=0;i<ht;i++){
    hipMalloc (( void **)& d_work[i] , sizeof ( hipComplex )* lwork );
	}
    int tupe_num= ht/PLAN1D_SIZE;
    int tupe_s= ht%PLAN1D_SIZE;
    if(tupe_num > 0){
    #pragma unroll
    for(int j=0;j<tupe_num;j++){
    #pragma unroll
	for(int i=0;i<PLAN1D_SIZE;i++){
       if( hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A+j*strid_A*PLAN1D_SIZE, lda, d_tau+i*tau+j*tau*PLAN1D_SIZE, d_work[i+j*PLAN1D_SIZE], lwork, devInfo[i]) !=HIPSOLVER_STATUS_SUCCESS){
		    fprintf(stdout,"[%s]:[%d] hipsolverDnCgeqrf error!",__FUNCTION__,__LINE__);
		    return;
       }
        }
     }
    #pragma unroll
	for(int i=0;i<tupe_s;i++){
        cusolver_status = hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A+tupe_num*strid_A*PLAN1D_SIZE, lda, d_tau+i*tau+tupe_num*tau*PLAN1D_SIZE, d_work[i+tupe_num*PLAN1D_SIZE], lwork, devInfo[i]);
	if(cusolver_status !=HIPSOLVER_STATUS_SUCCESS){
	fprintf(stderr,"[%s]:[%d]ERROR!",__FUNCTION__,__LINE__);
		    return;
	        }
        }
    }else{
    #pragma unroll
	for(int i=0;i<tupe_s;i++){
        cusolver_status = hipsolverDnCgeqrf(cusolverH[i], m, n, d_fftData+i*strid_A, lda, d_tau+i*tau, d_work[i], lwork, devInfo[i]);
	if(cusolver_status !=HIPSOLVER_STATUS_SUCCESS){
	fprintf(stderr,"[%s]:[%d]ERROR!",__FUNCTION__,__LINE__);
		    return;
	        }
    }
    }

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
    hipsolverDnDestroy(cusolverH[i]);
        }
	
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    hipMemcpy(Tau,d_tau,sizeof(hipComplex)* ht * tau,hipMemcpyDeviceToHost);
	//Tifft
	hipfftHandle iplan =0;
	in[0] = ht;
	on[0] = tupe;
	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	//printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));
	
	float* d_qr;
	hipMalloc((void**)&d_qr,sizeof(float)*tupe*bat);
	if(hipfftExecC2R(iplan,(hipfftComplex*)d_fftData,d_qr)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
    
    float* h_qr = (float*)malloc(tupe*bat*sizeof(float));
    hipMemcpy(h_qr,d_qr,sizeof(float)*tupe*bat,hipMemcpyDeviceToHost);
    for(int i=0;i<tupe*bat;i++){
	A[i]=h_qr[i]/tupe;
    }	
    free(h_qr);
    hipFree(d_fftData);
    hipFree(d_tau);
 
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){	
    hipFree(devInfo[i]);
    }
    free(devInfo);
    #pragma unroll
    for(int i=0;i<ht;i++){	
    hipFree(d_work[i]);
    }
    free(d_work);
    hipDeviceReset();
}
