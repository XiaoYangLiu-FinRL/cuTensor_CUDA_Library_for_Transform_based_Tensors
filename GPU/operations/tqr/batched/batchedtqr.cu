#include "hip/hip_runtime.h"
#include"batchedtqr.h"
void batchedtqr(float* t,const int m,const int n,const int tupe,hipComplex* tau)
{
	int ht  = tupe/2+1;
	int bat = m*n;
	float* d_t;
	hipfftComplex* d_fftData;
	hipMalloc((void**)&d_t,sizeof(float)*bat*tupe);
	hipMalloc((void**)&d_fftData,sizeof(hipfftComplex)*bat*ht);
	hipMemcpy(d_t,t,sizeof(float)*bat*tupe,hipMemcpyHostToDevice);

	//tff
	hipfftHandle plan;
	int n_f[1]   = {tupe};
	int stride = bat,dist = 1;
	int in[1]  = {tupe};
	int on[1]  = {ht};
	size_t worksize=0;
	if (hipfftPlanMany(&plan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_R2C,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(plan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_R2C,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
//	printf("the work size is:%lf G\n",(double)worksize/(1024*1024*1024));

	if(hipfftExecR2C(plan,d_t,(hipfftComplex*)d_fftData)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	if(d_t !=NULL){
	hipFree(d_t);
    d_t=NULL;   
    }
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d]cufftDestory faile!",__FUNCTION__,__LINE__);
		return;
	}

    if(magma_init() != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_init error!",__FUNCTION__,__LINE__);
		return;
    }
    magma_queue_t queue=NULL;
    magma_int_t dev = 0;
    magma_queue_create(dev, &queue);
    
//	    magmaFloatComplex *h_Amagma;
// 	    magmaFloatComplex *htau_magma;
    magmaFloatComplex *d_A, *dtau_magma;
    magmaFloatComplex **dA_array = NULL;
    magmaFloatComplex **dtau_array = NULL;

    magma_int_t   *dinfo_magma;
    magma_int_t M, N, lda, ldda, min_mn;
    magma_int_t batchCount;
    magma_int_t column;

    M = m;
    N = n;
    batchCount = ht;
    min_mn = min(M, N);
    lda    = M;
//            n2     = lda * N * batchCount;
//    ldda = ((M+31)/32)*32;
    ldda = magma_roundup( M, 32 );
//            magma_cmalloc_cpu( &h_Amagma,   n2     );
//            magma_cmalloc_cpu( &htau_magma, min_mn * batchCount );
     if(magma_cmalloc( &d_A,   ldda*N * batchCount ) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_malloc error!",__FUNCTION__,__LINE__);
		return;
     }

     if(magma_cmalloc( &dtau_magma,  min_mn * batchCount ) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_malloc error!",__FUNCTION__,__LINE__);
		return;
     }

     if(magma_imalloc( &dinfo_magma,  batchCount ) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_malloc error!",__FUNCTION__,__LINE__);
		return;
     }
 
     if(magma_malloc((void**) &dA_array,   batchCount * sizeof(magmaFloatComplex*) ) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_malloc error!",__FUNCTION__,__LINE__);
		return;
     }
     if(magma_malloc((void**) &dtau_array, batchCount * sizeof(magmaFloatComplex*) ) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_malloc error!",__FUNCTION__,__LINE__);
		return;
     }
     column = N * batchCount;

     magma_ccopymatrix(M, column, d_fftData, M, d_A, ldda, queue );
	
//   magma_cprint_gpu(M, column, d_fftData, M, queue );
//   magma_cprint_gpu(M, column, d_A, ldda, queue );
         
     magma_cset_pointer( dA_array, d_A, 1, 0, 0, ldda*N, batchCount, queue );
     magma_cset_pointer( dtau_array, dtau_magma, 1, 0, 0, min_mn, batchCount, queue );
  
//   magma_cprint_gpu(M, column, d_A, ldda, queue );

//   magma_cprint_gpu(M, column, d_fftData, M, queue );

    if( magma_cgeqrf_batched(M, N, dA_array, ldda, dtau_array, dinfo_magma, batchCount, queue) != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_cgeqrf_batched!",__FUNCTION__,__LINE__);
		return;
    }
//   magma_cprint_gpu(M, column, d_A, ldda, queue );
//         magma_cgetmatrix( M, column, d_A, ldda, h_Amagma, lda, queue );
//   magma_cgetmatrix(min_mn, batchCount, dtau_magma, min_mn, htau_magma, min_mn, queue );
     magma_cgetmatrix(min_mn, batchCount, dtau_magma, min_mn, tau, min_mn, queue );
     
//   magma_cprint( M, column, h_Amagma, lda);
//   magma_cprint(min_mn, batchCount, htau_magma, min_mn);

     magma_ccopymatrix(M, column, d_A, ldda, d_fftData, lda, queue );
     magma_queue_destroy( queue );
     if( d_A != NULL ){ 
     magma_free( d_A   );
     d_A = NULL;
     }
     if( dtau_magma != NULL ){
     magma_free( dtau_magma  );
     dtau_magma = NULL;
     }
     if( dinfo_magma != NULL){
     magma_free( dinfo_magma );
     dinfo_magma = NULL;
     }
     if( dA_array != NULL){
     magma_free( dA_array   );
     dA_array = NULL;
     }
     if( dtau_array != NULL){
     magma_free( dtau_array  );
     dtau_array = NULL;
     }
     if( magma_finalize() != MAGMA_SUCCESS){
		fprintf(stdout,"[%s]:[%d]magma_finalize error!",__FUNCTION__,__LINE__);
		return;
     }
	
	hipfftHandle iplan =0;
	in[0] = ht;
	on[0] = tupe;
	
	if (hipfftPlanMany(&iplan,1,n_f,in,stride,dist,on,stride,dist,
				HIPFFT_C2R,bat)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Plan creation failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	//estimate of the work size
	if(hipfftGetSizeMany(iplan,1,n_f,in,stride,dist,on,stride,dist,
			HIPFFT_C2R,bat,&worksize)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Estimate work size failed!",__FUNCTION__,__LINE__);
		return;
 	}
	//printf("the work size is:%ld G\n",(double)worksize/(1024*1024*1024));
	
	float* d_qr;
	hipMalloc((void**)&d_qr,sizeof(float)*tupe*bat);
	if(hipfftExecC2R(iplan,(hipfftComplex*)d_fftData,d_qr)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: Exec failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

    int num=bat*tupe;
	int threads,blocks;
    if(num<512){
        threads=num;
        blocks=1;
    }else{
	    threads=512;
	    blocks=((num%512 ==0)?num/512:num/512+1);
	}
        fftResultProcess<<<blocks,threads>>>(d_qr,num,tupe);

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}

	hipMemcpy(t,d_qr,sizeof(float)*bat*tupe,hipMemcpyDeviceToHost);
    if(d_qr !=NULL){
    hipFree(d_qr);
    d_qr=NULL;
    }
    if(d_fftData!=NULL){
	hipFree(d_fftData);
    d_fftData=NULL;
    }
hipDeviceReset();            
}
