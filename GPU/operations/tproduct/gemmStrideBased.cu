#include "tprod.h"
void gemmStrideBased(hipComplex* A, hipComplex* B, hipComplex* C, int Am, int An, int Ak, int Bn) {
    hipblasHandle_t handle;
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;
    int Bm = An;
    int Bk = Ak;
    //int strA = Am*An;
    //int strB = Bm*Bn;
    //int strC = Am*Bn;
    //int batchCount = Ak;

    hipComplex *d_A, *d_B, *d_C;
    hipMalloc ((void**)&d_A, sizeof(hipComplex) * Am*An*Ak);
    hipMalloc ((void**)&d_B, sizeof(hipComplex) * Bm*Bn*Bk);
    hipMalloc ((void**)&d_C, sizeof(hipComplex) * Am*Bn*Ak);

    hipMemcpy(d_A, A, sizeof(hipComplex) * Am*An*Ak, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(hipComplex) * Bm*Bn*Bk, hipMemcpyHostToDevice);

    for (int i=0; i<1; i++)
    {
    hipblasCreate(&handle);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am*Am, Bn*Am, Bm, &alpha, d_A, Am*Am, d_B, Bm, &beta, d_C, Am*Am);
    //hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm, &alpha, d_A+strA*i, Am, d_B+strB*i, Bm, &beta, d_C+strC*i, Am);
    }
    hipMemcpy(C, d_C, sizeof(hipComplex) * Am*Bn*Ak, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

