#include "tprod.h"
#include "based.h"
void streamedtprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe) {
    int ht = tupe/2 + 1;//half tupe
    int bat1 = row*rank;
    int bat2 = col*rank;
    int bat = bat1 + bat2;
    float *d_t;
    
    hipMalloc((void**)&d_t, tupe*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,ht*bat*sizeof(hipfftComplex));
    hipMemcpy(d_t,t1,tupe*bat1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_t+tupe*bat1, t2,tupe*bat2*sizeof(float),hipMemcpyHostToDevice);
    //tfft
   
    hipfftHandle plan =0;
    hipfftHandle plan2 =0;


    int n[1] = {tupe};
    int stride = bat1, dist = 1;
    int in[1] = {tupe};
    int on[1] = {ht};
    
    if(hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat1)==HIPFFT_ALLOC_FAILED) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if(hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
    
    int stride2 = bat2;
    if
        (hipfftPlanMany(&plan2,1, n, in, stride2, dist, on, stride2, dist,
                       HIPFFT_R2C, bat2)!=HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if
        (hipfftExecR2C(plan2, d_t+tupe*bat1,d_fftData+ht*bat1)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
    //destroy plan1 and plan2
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftDestroy(plan2)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
    hipFree(d_t);
	
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	//gemmbatched

	hipfftComplex* d_Tf;
 	hipMalloc((void**)&d_Tf,ht*row*col*sizeof(hipfftComplex));
	hipblasHandle_t* handle=(hipblasHandle_t *)malloc(PLAN1D_SIZE*sizeof(hipblasHandle_t));
    memset(handle,0,sizeof(hipblasHandle_t));
	hipComplex alpha;
	alpha.x =1;
	alpha.y =0;
	hipComplex beta;
	beta.x = 0;
	beta.y = 0;
	int Am = row;
	int An = rank;
	int Bn = col;
	int Bm = rank;
	int strA = Am*An;
	int strB = Bm*Bn;
	int strC = Am*Bn;
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipblasCreate(&handle[i]);
    hipblasSetStream(handle[i],stream[i]);
    }
    int tupe_num=ht/PLAN1D_SIZE;
    int tupe_s=ht%PLAN1D_SIZE;
    if(tupe_num > 0){
    #pragma unroll
    for(int j=0;j<tupe_num;j++){
    #pragma unroll
	for(int i=0; i<PLAN1D_SIZE; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA+strA*j*PLAN1D_SIZE, Am,d_fftData+strA*ht+i*strB+strB*j*PLAN1D_SIZE, Bm,  &beta,
	        d_Tf+i*strC+j*strC*PLAN1D_SIZE, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
        	}
	  }
    }
    #pragma unroll
	for(int i=0; i<tupe_s; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA+strA*tupe_num*PLAN1D_SIZE, Am,d_fftData+strA*ht+i*strB+strB*tupe_num*PLAN1D_SIZE, Bm,  &beta,
	        d_Tf+i*strC+tupe_num*strC*PLAN1D_SIZE, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	  }
    }else{
    #pragma unroll
	for(int i=0; i<tupe_s; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA, Am,d_fftData+strA*ht+i*strB, Bm,  &beta,
	        d_Tf+i*strC, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	  }
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipblasDestroy(handle[i]);
        hipStreamDestroy(stream[i]);
    }


	hipFree(d_fftData);

	//Tifft
    hipfftHandle iplan;
    in[0] = ht;
    on[0] = tupe;
    float* d_T;
    bat = row*col;
    stride = bat;
    hipMalloc((void**)&d_T, tupe*bat*sizeof(float));
    if
        (hipfftPlanMany(&iplan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)==HIPFFT_INTERNAL_ERROR) {
            fprintf(stdout, "[%s]:[%d]CUIFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if
        (hipfftExecC2R(iplan,(hipfftComplex*)d_Tf, d_T)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUIFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
    float* host_t = (float*)malloc(tupe*bat*sizeof(float));
    hipMemcpy(host_t,d_T,sizeof(float)*tupe*bat,hipMemcpyDeviceToHost);
	//transform
    for(int i=0;i<tupe*bat;i++){
	T[i]=host_t[i]/tupe;	
	}
//	for(int i=0;i<bat;i++){
//	  for(int j=0;j<tupe;j++){
//		T[j*bat+i]=host_t[i*tupe+j].x/tupe;
//		}
//	}

    if(stream != NULL){
    free(stream);
    stream=NULL;
    } 
   hipFree(d_Tf);
   free(host_t);
}
