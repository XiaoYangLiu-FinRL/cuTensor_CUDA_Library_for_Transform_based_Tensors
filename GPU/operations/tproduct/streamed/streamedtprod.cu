#include "tprod.h"
#include "based.h"
void streamedtprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe) {
	int bat = row*rank;
	int bat2 = rank*col;
//	hipfftComplex* t_f = (hipfftComplex*)malloc(bat*tupe*sizeof(hipfftComplex));
//	hipfftComplex* t_f2 = (hipfftComplex*)malloc(bat2*tupe*sizeof(hipfftComplex));
    hipfftComplex* t_f;
    hipfftComplex* t_f2;

    hipHostAlloc((void**)&t_f,bat*tupe*sizeof(hipfftComplex),hipHostMallocDefault);
    hipHostAlloc((void**)&t_f2,bat2*tupe*sizeof(hipfftComplex),hipHostMallocDefault);
    

    //transform t1
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t1[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}

	//transform t2
	for(int i=0;i<bat2;i++){
	   for(int j=0;j<tupe;j++){
		t_f2[i*tupe+j].x=t2[j*bat2+i];
		t_f2[i*tupe+j].y=0;
		}
	}
    //set stream for t1 and t2
    hipStream_t* stream = (hipStream_t*)malloc(PLAN1D_SIZE*sizeof(hipStream_t));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamCreate(&stream[i]);
    }

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipfftComplex* d_fftData2;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
	hipMalloc((void**)&d_fftData2,tupe*bat2*sizeof(hipfftComplex));
    //process bat
    int bat_num = bat/PLAN1D_SIZE;
    int bat_s = bat%PLAN1D_SIZE;
	hipfftHandle * plan=(hipfftHandle*)malloc(sizeof(hipfftHandle)*PLAN1D_SIZE);
    memset(plan,0,sizeof(hipfftHandle));
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&plan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipfftSetStream(plan[i],stream[i]);
    }
    if(bat_num > 0){
    for(int j=0;j<bat_num;j++){

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,t_f+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
     }

    #pragma unroll	
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
            	}
        	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(t_f+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
         }
    }

    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+PLAN1D_SIZE*bat_num*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe+bat_num*tupe*PLAN1D_SIZE,d_fftData+i*tupe+tupe*bat_num*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }else{
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(d_fftData+i*tupe,t_f+i*tupe,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat_s;i++){
	if(hipfftExecC2C(plan[i],d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	    }
	}
    #pragma unroll
    for(int i=0;i<bat_s;i++){
	hipMemcpyAsync(t_f+i*tupe,d_fftData+i*tupe,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }

    //process bat2
    int bat2_s=bat2%PLAN1D_SIZE;
    int bat2_num=bat2/PLAN1D_SIZE;
	hipfftHandle * plan2=(hipfftHandle*)malloc(sizeof(hipfftHandle)*PLAN1D_SIZE);
    memset(plan2,0,sizeof(hipfftHandle));

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&plan2[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipfftSetStream(plan2[i],stream[i]);
    }

    if(bat2_num > 0){
    for(int j=0;j<bat2_num;j++){

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(d_fftData2+i*tupe+j*tupe*PLAN1D_SIZE,t_f2+i*tupe+j*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
     }

    #pragma unroll	
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(plan2[i],d_fftData2+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData2+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
            	}
        	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(t_f2+i*tupe+j*tupe*PLAN1D_SIZE,d_fftData2+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
         }
    }

    #pragma unroll
    for(int i=0;i<bat2_s;i++){
	hipMemcpyAsync(d_fftData2+i*tupe+bat2_num*tupe*PLAN1D_SIZE,t_f2+i*tupe+bat2_num*tupe*PLAN1D_SIZE,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat2_s;i++){
	if(hipfftExecC2C(plan2[i],d_fftData2+i*tupe+bat2_num*tupe*PLAN1D_SIZE,d_fftData2+i*tupe+PLAN1D_SIZE*bat2_num*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat2_s;i++){
	hipMemcpyAsync(t_f2+i*tupe+bat2_num*tupe*PLAN1D_SIZE,d_fftData2+i*tupe+tupe*bat2_num*PLAN1D_SIZE,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }else{
    #pragma unroll
    for(int i=0;i<bat2_s;i++){
	hipMemcpyAsync(d_fftData2+i*tupe,t_f2+i*tupe,tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice,stream[i]);
    }
    #pragma unroll	
	for(int i=0;i<bat2_s;i++){
	if(hipfftExecC2C(plan2[i],d_fftData2+i*tupe,d_fftData2+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<bat2_s;i++){
	hipMemcpyAsync(t_f2+i*tupe,d_fftData2+i*tupe,sizeof(hipfftComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }

	//transform
    hipfftComplex* t_f3 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);
	hipfftComplex* t_f4 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat2);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f3[j*bat+i]=t_f[i*tupe+j];
		}
	}
	for(int i=0;i<bat2;i++){
	  for(int j=0;j<tupe;j++){
		t_f4[j*bat2+i]=t_f2[i*tupe+j];
		}
	}
	hipMemcpy(d_fftData,t_f3,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);
	hipMemcpy(d_fftData2,t_f4,sizeof(hipfftComplex)*bat2*tupe,hipMemcpyHostToDevice);
    //destroy plan1 and plan2
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftDestroy(plan[i])!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftDestroy(plan2[i])!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
    }
	if(t_f != NULL){
	hipHostFree(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	hipHostFree(t_f2);
	t_f2 = NULL;	
	}
	if(t_f3 != NULL){
	free(t_f3);
	t_f3 = NULL;
	}
	if(t_f4 != NULL){
	free(t_f4);
	t_f4 = NULL;
	}
    if(plan != NULL){
    free(plan);
    plan=NULL;
    }
    if(plan2 != NULL){
    free(plan2);
    plan2=NULL;
    }

	//gemmbatched

	hipfftComplex* d_Tf;
 	hipMalloc((void**)&d_Tf,tupe*row*col*sizeof(hipfftComplex));
	hipblasHandle_t* handle=(hipblasHandle_t *)malloc(PLAN1D_SIZE*sizeof(hipblasHandle_t));
    memset(handle,0,sizeof(hipblasHandle_t));
	hipComplex alpha;
	alpha.x =1;
	alpha.y =0;
	hipComplex beta;
	beta.x = 0;
	beta.y = 0;
	int Am = row;
	int An = rank;
	int Bn = col;
	int Bm = rank;
	int strA = Am*An;
	int strB = Bm*Bn;
	int strC = Am*Bn;
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipblasCreate(&handle[i]);
    hipblasSetStream(handle[i],stream[i]);
    }
    int tupe_num=tupe/PLAN1D_SIZE;
    int tupe_s=tupe%PLAN1D_SIZE;
    if(tupe_num > 0){
    #pragma unroll
    for(int j=0;j<tupe_num;j++){
    #pragma unroll
	for(int i=0; i<PLAN1D_SIZE; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA+strA*j*PLAN1D_SIZE, Am,d_fftData2+i*strB+strB*j*PLAN1D_SIZE, Bm,  &beta,
	        d_Tf+i*strC+j*strC*PLAN1D_SIZE, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
        	}
	  }
    }
    #pragma unroll
	for(int i=0; i<tupe_s; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA+strA*tupe_num*PLAN1D_SIZE, Am,d_fftData2+i*strB+strB*tupe_num*PLAN1D_SIZE, Bm,  &beta,
	        d_Tf+i*strC+tupe_num*strC*PLAN1D_SIZE, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	  }
    }else{
    #pragma unroll
	for(int i=0; i<tupe_s; i++){
	if(hipblasCgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA, Am,d_fftData2+i*strB, Bm,  &beta,
	        d_Tf+i*strC, Am) !=HIPBLAS_STATUS_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	  }
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipblasDestroy(handle[i]);
    }


	hipFree(d_fftData);
	hipFree(d_fftData2);

	//Tifft

	hipComplex* host_result=(hipComplex*)malloc(sizeof(hipComplex)*tupe*row*col);
	hipComplex* host_result2;

    hipHostAlloc((void**)&host_result2,sizeof(hipComplex)*tupe*row*col,hipHostMallocDefault);

	hipMemcpy(host_result,d_Tf,sizeof(hipComplex)*tupe*row*col,hipMemcpyDeviceToHost);

	//transform
	int bat3=row*col;
	for(int i=0;i<bat3;i++){
	  for(int j=0;j<tupe;j++){
		host_result2[i*tupe+j]=host_result[j*bat3+i];
		}
	}
	hipfftHandle* iplan = (hipfftHandle*)malloc(PLAN1D_SIZE*sizeof(hipfftHandle));
    memset(iplan,0,sizeof(hipfftHandle));

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftPlan1d(&iplan[i],tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
    hipfftSetStream(iplan[i],stream[i]);
    }
	
	//ifft
    int bat3_num=bat3/PLAN1D_SIZE;
    int bat3_s=bat3%PLAN1D_SIZE;
    if(bat3_num > 0){
    #pragma unroll
    for(int j=0;j<bat3_num;j++){
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(d_Tf+i*tupe+j*tupe*PLAN1D_SIZE,host_result2+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
	for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftExecC2C(iplan[i],d_Tf+i*tupe+j*tupe*PLAN1D_SIZE,d_Tf+i*tupe+j*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	hipMemcpyAsync(host_result2+i*tupe+j*tupe*PLAN1D_SIZE,d_Tf+i*tupe+j*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
    }
    }
    #pragma unroll
    for(int i=0;i<bat3_s;i++){
	hipMemcpyAsync(d_Tf+i*tupe+bat3_num*tupe*PLAN1D_SIZE,host_result2+i*tupe+bat3_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    #pragma unroll
	for(int i=0;i<bat3_s;i++){
	if(hipfftExecC2C(iplan[i],d_Tf+i*tupe+bat3_num*tupe*PLAN1D_SIZE,d_Tf+i*tupe+bat3_num*tupe*PLAN1D_SIZE,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
        	}
    }
    #pragma unroll
    for(int i=0;i<bat3_s;i++){
	hipMemcpyAsync(host_result2+i*tupe+bat3_num*tupe*PLAN1D_SIZE,d_Tf+i*tupe+bat3_num*tupe*PLAN1D_SIZE,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
          }

    }else{

    #pragma unroll
    for(int i=0;i<bat3_s;i++){
	hipMemcpyAsync(d_Tf+i*tupe,host_result2+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyHostToDevice,stream[i]);
	}
    #pragma unroll
	for(int i=0;i<bat3_s;i++){
	if(hipfftExecC2C(iplan[i],d_Tf+i*tupe,d_Tf+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
        	}
    }
    #pragma unroll
    for(int i=0;i<bat3_s;i++){
	hipMemcpyAsync(host_result2+i*tupe,d_Tf+i*tupe,sizeof(hipComplex)*tupe,hipMemcpyDeviceToHost,stream[i]);
          }
	}

    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
        hipStreamSynchronize(stream[i]);
    }
    #pragma unroll
    for(int i=0;i<PLAN1D_SIZE;i++){
	if(hipfftDestroy(iplan[i])!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
        hipStreamDestroy(stream[i]);
    }
	//transform
	for(int i=0;i<bat3;i++){
	  for(int j=0;j<tupe;j++){
		T[j*bat3+i]=host_result2[i*tupe+j].x/tupe;
		}
	}

    if(stream != NULL){
    free(stream);
    stream=NULL;
    } 
	if(host_result != NULL){
	free(host_result);
	host_result = NULL;
	}

	if(host_result2 != NULL){
	hipHostFree(host_result2);
	host_result2 = NULL;
	}

	hipFree(d_Tf);
}
