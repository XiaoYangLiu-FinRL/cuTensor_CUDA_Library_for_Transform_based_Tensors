#include "tprod.h"
#include "based.h"
void basedtprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe) {
	int bat = row*rank;
	int bat2 = rank*col;
	hipfftComplex* t_f = (hipfftComplex*)malloc(bat*tupe*sizeof(hipfftComplex));
	hipfftComplex* t_f2 = (hipfftComplex*)malloc(bat2*tupe*sizeof(hipfftComplex));
	//transform t1
	for(int i=0;i<bat;i++){
	   for(int j=0;j<tupe;j++){
		t_f[i*tupe+j].x=t1[j*bat+i];
		t_f[i*tupe+j].y=0;
		}
	}

	//transform t2
	for(int i=0;i<bat2;i++){
	   for(int j=0;j<tupe;j++){
		t_f2[i*tupe+j].x=t2[j*bat2+i];
		t_f2[i*tupe+j].y=0;
		}
	}

	//tfft:C2C
	hipfftComplex* d_fftData;
	hipfftComplex* d_fftData2;
	hipMalloc((void**)&d_fftData,tupe*bat*sizeof(hipfftComplex));	
	hipMalloc((void**)&d_fftData2,tupe*bat2*sizeof(hipfftComplex));	
	hipMemcpy(d_fftData,t_f,bat*tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipMemcpy(d_fftData2,t_f2,bat2*tupe*sizeof(hipfftComplex),hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftHandle plan2;

	if(hipfftPlan1d(&plan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(hipfftPlan1d(&plan2,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
	 	fprintf(stdout,"[%s]:[%d] hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	for(int i=0;i<bat;i++){
	if(hipfftExecC2C(plan,d_fftData+i*tupe,d_fftData+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	for(int i=0;i<bat2;i++){
	if(hipfftExecC2C(plan2,d_fftData2+i*tupe,d_fftData2+i*tupe,HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda syncthronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//transform
	hipMemcpy(t_f,d_fftData,sizeof(hipfftComplex)*bat*tupe,hipMemcpyDeviceToHost);
	hipMemcpy(t_f2,d_fftData2,sizeof(hipfftComplex)*bat2*tupe,hipMemcpyDeviceToHost);
	hipfftComplex* t_f3 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat);
	hipfftComplex* t_f4 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*tupe*bat2);

	for(int i=0;i<bat;i++){
	  for(int j=0;j<tupe;j++){
		t_f3[j*bat+i]=t_f[i*tupe+j];
		}
	}
	for(int i=0;i<bat2;i++){
	  for(int j=0;j<tupe;j++){
		t_f4[j*bat2+i]=t_f2[i*tupe+j];
		}
	}
	
	hipMemcpy(d_fftData,t_f3,sizeof(hipfftComplex)*bat*tupe,hipMemcpyHostToDevice);
	hipMemcpy(d_fftData2,t_f4,sizeof(hipfftComplex)*bat2*tupe,hipMemcpyHostToDevice);
	
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}

	if(hipfftDestroy(plan2)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}
		
	if(t_f != NULL){
	free(t_f);
	t_f = NULL;
	}
	if(t_f2 !=NULL){
	free(t_f2);
	t_f2 = NULL;	
	}
	if(t_f3 != NULL){
	free(t_f3);
	t_f3 = NULL;
	}
	if(t_f4 != NULL){
	free(t_f4);
	t_f4 = NULL;
	}
	//gemmbatched

	hipfftComplex* d_Tf;
 	hipMalloc((void**)&d_Tf,tupe*row*col*sizeof(hipfftComplex));
	hipblasHandle_t handle;
	hipComplex alpha;
	alpha.x =1;
	alpha.y =0;
	hipComplex beta;
	beta.x = 0;
	beta.y = 0;
	int Am = row;
	int An = rank;
	int Bn = col;
	int Bm = rank;
	int strA = Am*An;
	int strB = Bm*Bn;
	int strC = Am*Bn;
	hipblasCreate(&handle);
	for(int i=0; i<tupe; i++){
	if(hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_fftData+i*strA, Am,d_fftData2+i*strB, Bm,  &beta,
	        d_Tf+i*strC, Am) !=HIPBLAS_STATUS_SUCCESS){
	
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	  }
	hipblasDestroy(handle);
		
	hipFree(d_fftData);
	hipFree(d_fftData2);
	//Tifft

	hipComplex* host_result=(hipComplex*)malloc(sizeof(hipComplex)*tupe*row*col);
	hipComplex* host_result2=(hipComplex*)malloc(sizeof(hipComplex)*tupe*row*col);

	hipMemcpy(host_result,d_Tf,sizeof(hipComplex)*tupe*row*col,hipMemcpyDeviceToHost);

	//transform
	int bat3=row*col;
	for(int i=0;i<bat3;i++){
	  for(int j=0;j<tupe;j++){
		host_result2[i*tupe+j]=host_result[j*bat3+i];
		}
	}
	hipMemcpy(d_Tf,host_result2,sizeof(hipComplex)*tupe*row*col,hipMemcpyHostToDevice);
	
	if(host_result != NULL){
	free(host_result);
	host_result = NULL;
	}

	if(host_result2 != NULL){
	free(host_result2);
	host_result2 = NULL;
	}

	hipfftHandle iplan;

	if(hipfftPlan1d(&iplan,tupe,HIPFFT_C2C,1) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipfftPlan1d failed!",__FUNCTION__,__LINE__);
		return;	
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	//ifft
	for(int i=0;i<bat3;i++){
	if(hipfftExecC2C(iplan,d_Tf+i*tupe,d_Tf+i*tupe,HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR:hipfftExecC2C failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	

	if(hipDeviceSynchronize() != hipSuccess){
		fprintf(stdout,"[%s]:[%d] cuda synchronize err!",__FUNCTION__,__LINE__);
		return;
	}
	
	if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){
		fprintf(stdout,"[%s]:[%d] hipfftDestroy failed!",__FUNCTION__,__LINE__);
		return;
	}

	hipComplex* host_T =(hipComplex*)malloc(sizeof(hipComplex)*tupe*row*col);
	hipMemcpy(host_T,d_Tf,sizeof(hipComplex)*tupe*bat3,hipMemcpyDeviceToHost);
	//transform

	for(int i=0;i<bat3;i++){
	  for(int j=0;j<tupe;j++){
		T[j*row*col+i]=host_T[i*tupe+j].x/tupe;
		}
	}

	free(host_T);
	hipFree(d_Tf);
}
