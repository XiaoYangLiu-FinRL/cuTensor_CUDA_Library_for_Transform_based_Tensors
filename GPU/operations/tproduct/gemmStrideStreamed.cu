//#include "gemmStrideStreamed.h"
#include"tprod.h"

void gemmStrideStreamed(hipComplex *A, hipComplex *B, hipComplex *C, int m, int n, int k, int l)
{
    //Allocate device memory for A B C
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m*n*k*sizeof(hipComplex));
    hipMalloc((void**)&d_B, n*l*k*sizeof(hipComplex));
    hipMalloc((void**)&d_C, m*l*k*sizeof(hipComplex));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    //transfer A B to device memory
    hipMemcpy(d_A, A, sizeof(hipComplex)  *m*n*k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(hipComplex)  *n*l*k, hipMemcpyHostToDevice);

    hipStream_t *streams = (hipStream_t *) malloc(k*sizeof(hipStream_t));
    for (int i=0; i<k; i++)
        hipStreamCreate(&streams[i]);
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;

    for (int i=0; i<k; i++)
    {
        hipblasSetStream(handle, streams[i]);

        hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, l, n, &alpha,
                d_A+i*m*n, m, d_B+i*n*l, n, &beta, d_C+i*m*l, m);
    }

    hipMemcpy(C, d_C, sizeof(hipComplex)*m*l*k, hipMemcpyDeviceToHost);
    for (int i=0; i<k; i++)
        hipStreamDestroy(streams[i]);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(streams);
    hipblasDestroy(handle);
}
