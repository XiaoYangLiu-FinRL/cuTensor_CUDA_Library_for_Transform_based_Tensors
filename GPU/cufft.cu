#include "fft.h"
void cufft(float *t,int l,int bat,hipfftComplex *tf)
{
	hipfftComplex *t_f = new hipfftComplex[l*bat];
//transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j].x=t[j*bat+i];
        t_f[i*l+j].y=0;
      }
    hipfftComplex *d_fftData;
    // hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    CHECK(hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex)));
    // if (hipGetLastError() != hipSuccess){
    //   printf(stderr, "Cuda error: Failed to allocate\n");
    //   return; 
    // }

    CHECK(hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice));

    for (int i=0; i<bat; i++)
    {
            hipfftHandle plan;
            CHECK_CUFFT(hipfftPlan1d(&plan,l,HIPFFT_C2C,1));
            CHECK_CUFFT(hipfftExecC2C(plan,d_fftData + l*i,d_fftData + l*i,HIPFFT_FORWARD));
            hipDeviceSynchronize();
            hipfftDestroy(plan);
    }
    CHECK(hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    // hipFree(d_fftData);
    hipFree(d_fftData);
//transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            tf[j*bat+i]=t_f[i*l+j];
          }
    delete[] t_f;
}

