#include "tprod.h"

void tprod(float* t1,float* t2,float* T,int row, int col, int rank, int tupe)
{
  hipfftComplex *t1f = new hipfftComplex[row*rank*tupe];
  hipfftComplex *t2f = new hipfftComplex[rank*col*tupe];
  Tfft(t1,tupe,row*rank,t1f);
  Tfft(t2,tupe,rank*col,t2f);
  hipfftComplex *Tf = new hipfftComplex[row*col*tupe];
  for(int i=0;i<row*col*tupe;i++){
    Tf[i].x=0;
    Tf[i].y=0;
  }
  for(int i=0; i<tupe;i++){
    for(int j=0;j<row;j++){
      for(int k=0;k<col;k++){
        for(int w=0;w<rank;w++){
          mul_cufft(t1f+i*row*rank+w*row+j,t2f+i*rank*col+k*rank+w,Tf+i*row*col+k*row+j);
        }
      }
    }
  }
  delete[] t1f;
  t1f = nullptr;
  delete[] t2f;
  t2f = nullptr;
  Tifft(T,tupe,row*col,Tf);
  delete[] Tf;
  Tf = nullptr;
}