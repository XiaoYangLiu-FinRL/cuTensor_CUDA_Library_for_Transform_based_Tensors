#include "hip/hip_runtime.h"
#include "cuNorm.h"
void cuNorm(int l)
{
    int n = l*l*l;
    double *v = new double[n];
    for (int i=0; i<n; i++)
        v[i] = random(1000);
    clock_t start, finish;
    hipblasHandle_t handle ;
    int incx = 1;
    double result;
    double *d_v ;
    start = clock();
    hipblasCreate(&handle);
    hipMalloc ((void**)&d_v, sizeof(double) * n);

    hipMemcpy(d_v, v, sizeof(double) * n, hipMemcpyHostToDevice);
    
    hipblasDnrm2(handle, n, d_v, incx, &result);
    hipFree(d_v);
    finish = clock();
    double time = (double)(finish-start) / CLOCKS_PER_SEC; 
    delete v;
    printf("result is : %lf\n", result);
    printf("time is : %lf\n", time);
}
void cuNorm2(int l)
{
    int n = l*l*l;
    double *v = new double[n];
    for (int i=0; i<n; i++)
        v[i] = random(1000);
    clock_t start, finish;
    hipblasHandle_t handle ;
    int incx = 1;
    double result;
    double *d_v ;
    start = clock();
    hipMalloc ((void**)&d_v, sizeof(double) * n);

    hipMemcpy(d_v, v, sizeof(double) * n, hipMemcpyHostToDevice);
    
    for (int j=0; j<l; j++)
    {
            hipblasCreate(&handle);
            hipblasDnrm2(handle, l*l, d_v+l*l*j, incx, &result);
    }
    hipFree(d_v);
    finish = clock();
    double time = (double)(finish-start) / CLOCKS_PER_SEC; 
    delete v;
    printf("result is : %lf\n", result);
    printf("time is : %lf\n", time);
}
