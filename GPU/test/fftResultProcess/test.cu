#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"based.h"
/*__global__ void fftResultProcess(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}*/
int main(){
	int num=100;
	int len=2;
	float* data = new float[num];
	float* out = new float[num];
	for(int i=0;i<num;i++){
	data[i]=i;	
	} 
	float* d_data;
	hipMalloc((void**)&d_data,sizeof(float)*num);
	hipMemcpy(d_data,data,sizeof(float)*num,hipMemcpyHostToDevice);
	int threads=0;
	int blocks=0;
	if(num<512){
	threads=num;
	blocks=1;
	}else{
	threads=512;
	blocks=(num%512 ==0)?num/512:num/512+1;
	}
	fftResultProcess<<<blocks,threads>>>(d_data,num,len);
	hipMemcpy(out,d_data,sizeof(float)*num,hipMemcpyDeviceToHost);
	for(int i=0;i<num;i++){
	std::cout<<out[i]<<std::endl;
	}
	return 0;
	}

