#include "mprod.h"
void mprod(hipComplex* A, hipComplex* B, hipComplex* C, int Am, int An, int Bn) {
    hipblasHandle_t handle;
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;
    int Bm = An;

    hipComplex *d_A, *d_B, *d_C;
    hipMalloc ((void**)&d_A, sizeof(hipComplex) * Am*An);
    hipMalloc ((void**)&d_B, sizeof(hipComplex) * Bm*Bn);
    hipMalloc ((void**)&d_C, sizeof(hipComplex) * Am*Bn);

    hipMemcpy(d_A, A, sizeof(hipComplex) * Am*An, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(hipComplex) * Bm*Bn, hipMemcpyHostToDevice);

   if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
	fprintf(stdout,"[%s][%d] hipblasCreate faile!",__FUNCTION__,__LINE__);
	return;
	}
    if(hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm, &alpha, d_A, Am, d_B, Bm, &beta, d_C, Am) !=HIPBLAS_STATUS_SUCCESS){
	fprintf(stdout,"[%s]:[%d] hipblasCgemm faile!",__FUNCTION__,__LINE__);
	return;
	
	}
    
    hipMemcpy(C, d_C, sizeof(hipComplex) * Am*Bn, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

