#include "Tfft.h"


void Tfft(float *t,int l,int bat,hipfftComplex *tf)
{
	hipfftComplex *t_f = new hipfftComplex[l*bat];
//transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j].x=t[j*bat+i];
        t_f[i*l+j].y=0;
      }
    hipfftComplex *d_fftData;
    // hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    CHECK(hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex)));
    // if (hipGetLastError() != hipSuccess){
    //   printf(stderr, "Cuda error: Failed to allocate\n");
    //   return; 
    // }

    CHECK(hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle plan =0;
    CHECK_CUFFT(hipfftPlan1d(&plan,l,HIPFFT_C2C,bat));
    CHECK_CUFFT(hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_FORWARD));
    hipDeviceSynchronize();
    CHECK(hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    hipfftDestroy(plan);
    // hipFree(d_fftData);
    hipFree(d_fftData);
//transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            tf[j*bat+i]=t_f[i*l+j];
          }
    delete[] t_f;
}
void mul_cufft(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c){
  c->x += a->x*b->x - a->y*b->y;
  c->y += a->x*b->y + a->y*b->x;
}
void transform(int a,int b,int c,hipfftComplex *t,hipfftComplex *tt)
{
  for(int i=0;i<c;i++)
    for(int j=0;j<a;j++)
      for(int k=0;k<b;k++){
        tt[i*a*b+j*b+k].x = t[i*a*b+k*a+j].x;
        tt[i*a*b+j*b+k].y = 0 - t[i*a*b+k*a+j].y;
      }
}
void Tifft(float *t,int l,int bat,hipfftComplex *tf)
{
  //to be update;use stream?
  hipfftComplex *t_f = new hipfftComplex[l*bat];
//transform
    for(int i=0;i<bat;i++)
      for(int j=0;j<l;j++){
        t_f[i*l+j]=tf[j*bat+i];
      }
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,l*bat*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,t_f,l*bat*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan =0;
    hipfftPlan1d(&plan,l,HIPFFT_C2C,bat);
    hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    hipMemcpy(t_f,d_fftData,l*bat*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_fftData);
//transform
    for(int i=0;i<bat;i++)
          for(int j=0;j<l;j++){
            t[j*bat+i]=t_f[i*l+j].x/l;
          }
    delete[] t_f;
}
void printTensor(int m, int n,int k, const float*A)
{
    for(int bt=0;bt<k;bt++){
      for(int row = 0 ; row < m ; row++){
          for(int col = 0 ; col < n ; col++){
              cout<<A[bt*m*n+row + col*m]<<" ";
          }
          cout<<endl;
      }
      cout<<"____________"<<endl;
    }
}
 void TprintTensor(int m, int n,int k, const hipfftComplex *A)
{
    for(int bt=0;bt<k;bt++){
      for(int row = 0 ; row < m ; row++){
          for(int col = 0 ; col < n ; col++){
              cout<<A[bt*m*n+row + col*m].x<<"+"<<A[bt*m*n+row + col*m].y<<" ";
          }
          cout<<endl;
      }
      cout<<"____________"<<endl;
    }
}
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}
