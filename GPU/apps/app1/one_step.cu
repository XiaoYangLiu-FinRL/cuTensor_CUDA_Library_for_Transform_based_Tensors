#include "one_step.h"
#include "Tfft.h"
void one_step(hipfftComplex* T_omega_f, hipfftComplex* omega_f, hipfftComplex* X_f, hipfftComplex* Y_f, int m, int n, int k,int r_)
{
    hipComplex *tensor_V = new hipComplex[k*m];
    hipComplex *temp = new hipComplex[k*m*r_*k];

    for(int i = 0; i < n; i++){
   
        
          for(int it=0;it<k*m*r_*k;it++){
            temp[it].x=0;
            temp[it].y=0;
        }
//        double tempS = cpuSecond();
        for(int ri = 0; ri < r_; ri++)
            for(int k2 = 0; k2 < k; k2++)
                for(int k1 = 0; k1 < k; k1++)
                    for(int mi =0; mi < m; mi++){
                        int col = r_ * k2 + ri;
                        int row = ((k1 + k2)%k) * m + mi;
                        mul_cufft(omega_f + k1*m*n + i*m + mi, X_f +k2*m*r_ + ri*m + mi,temp + col*m*k + row);
                        // temp[col*m*k + row] = t[k1*m*n + mi] * t[k2*m*n + ri*m + mi];
                    }
        // cout << "temp " <<i << endl;
        // TprintTensor(k*m,k*r_,1,temp);
       /* double tempE = cpuSecond() - tempS;
        printf("Time of temp %f sec\n", tempE);
       */
        for(int j = 0; j < k; j++)
            for(int it = 0; it < m;it++)
                tensor_V[j*m + it]=T_omega_f[j*m*n + i*m + it];
        // cout << "tensor_V" << endl;
        // TprintTensor(k, m, 1,tensor_V); 
  //      double qrS = cpuSecond();  
        hipsolverHandle_t cusolverH = NULL;
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
        hipError_t cudaStat1 = hipSuccess;
        hipError_t cudaStat2 = hipSuccess;
        hipError_t cudaStat3 = hipSuccess;
        hipError_t cudaStat4 = hipSuccess;

        hipComplex *d_A = NULL; // linear memory of GPU  
        hipComplex *d_tau = NULL; // linear memory of GPU 
        hipComplex *d_B  = NULL; 
        int *devInfo = NULL; // info in gpu (device copy)
        hipComplex *d_work = NULL;
        int  lwork = 0; 

//        int info_gpu = 0;

        hipComplex one;
        one.x = 1;
        one.y =0;

        // step 1: create cusolver/cublas handle
        cusolver_status = hipsolverDnCreate(&cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        // step 2: copy A and B to device
        cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(hipComplex) * m*k*r_*k);
        cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(hipComplex) * r_*k);
        cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(hipComplex) * m*k);
        cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
        assert(hipSuccess == cudaStat1);
        assert(hipSuccess == cudaStat2);
        assert(hipSuccess == cudaStat3);
        assert(hipSuccess == cudaStat4);

        cudaStat1 = hipMemcpy(d_A, temp, sizeof(hipComplex) *m*k*r_*k, hipMemcpyHostToDevice);
        cudaStat2 = hipMemcpy(d_B, tensor_V, sizeof(hipComplex) * m*k, hipMemcpyHostToDevice);
        assert(hipSuccess == cudaStat1);
        assert(hipSuccess == cudaStat2);

        // step 3: query working space of geqrf and ormqr
        cusolver_status = hipsolverDnCgeqrf_bufferSize(
            cusolverH, 
            m*k, 
            k*r_, 
            d_A, 
            m*k, 
            &lwork);
        assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

        cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipComplex)*lwork);
        assert(hipSuccess == cudaStat1);

        // step 4: compute QR factorization
        cusolver_status = hipsolverDnCgeqrf(
            cusolverH, 
            m*k, 
            k*r_, 
            d_A, 
            m*k, 
            d_tau, 
            d_work, 
            lwork, 
            devInfo);
        cudaStat1 = hipDeviceSynchronize();
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
        assert(hipSuccess == cudaStat1);

        // check if QR is good or not
        // cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
        // assert(hipSuccess == cudaStat1);

        // printf("after geqrf: info_gpu = %d\n", info_gpu);
        // assert(0 == info_gpu);

        // step 5: compute Q^T*B
        cusolver_status= hipsolverDnCunmqr(
            cusolverH, 
            HIPBLAS_SIDE_LEFT, 
            HIPBLAS_OP_C,
            m*k, 
            1, 
            k*r_, 
            d_A, 
            m*k,
            d_tau,
            d_B,
            m*k,
            d_work,
            lwork,
            devInfo);
        cudaStat1 = hipDeviceSynchronize();
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
        assert(hipSuccess == cudaStat1);
        // check if QR is good or not
        // cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
        // assert(hipSuccess == cudaStat1);

        // printf("after ormqr: info_gpu = %d\n", info_gpu);
        // assert(0 == info_gpu);
// step 6: compute x = R \ Q^T*B

        cublas_status = hipblasCtrsm(
            cublasH,
            HIPBLAS_SIDE_LEFT,
            HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N, 
            HIPBLAS_DIAG_NON_UNIT,
            k*r_,
            1,
            &one,
            d_A,
            m*k,
            d_B,
            m*k);
        cudaStat1 = hipDeviceSynchronize();
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        assert(hipSuccess == cudaStat1);
        hipComplex temp_Y_f[r_*k];
        cudaStat1 = hipMemcpy(temp_Y_f, d_B, sizeof(hipComplex)*r_*k, hipMemcpyDeviceToHost);
        assert(hipSuccess == cudaStat1);
        hipMemcpy(temp, d_A, sizeof(hipComplex)*m*k, hipMemcpyDeviceToHost);

        if (d_A    ) hipFree(d_A);
        if (d_tau  ) hipFree(d_tau);
        if (d_B    ) hipFree(d_B);
        if (devInfo) hipFree(devInfo);
        if (d_work ) hipFree(d_work);


        if (cublasH ) hipblasDestroy(cublasH);   
        if (cusolverH) hipsolverDnDestroy(cusolverH);   
        // hipDeviceReset();
    //     Y_f(:,i,j) = temp_Y_f((j-1)*r + 1 : j*r);
    // end
        // cout << "temp_Y_f" << endl;
        // TprintTensor(r_, k, 1,temp_Y_f);
        // cout << "temp" << endl;
        // TprintTensor(k*m,k*r_,1,temp);
/*        double qrE = cpuSecond() - qrS;
        printf("Time of QR %f sec\n", qrE);
  */
        for (int j = 0; j < k; j++)
        	for (int a =0; a < r_; a++)
        		Y_f[j*r_*n + i*r_ + a] = temp_Y_f[j*r_ + a];
    }
    delete[] temp;
    temp = NULL;
    delete[] tensor_V;
    tensor_V = NULL;
}
