#include "hip/hip_runtime.h"
#include"sol_v.h"
/**
* solve tensor US*V=T
* US is a tensor , size: m * min(m,n) * k
* T is a tensor , size: m * n * k
* V is a tensor ,size: min(m,n) * n * k 
*/
void solve_v(float* T,float* US,const int m,const int n,const int k,float* V){
int hk = k/2+1;
int min_val = Min(m,n);
hipComplex *tf, *uf, *vf;
tf = (hipComplex*)malloc( sizeof(hipComplex)* m* n* hk);
uf = (hipComplex*)malloc( sizeof(hipComplex)* m * min_val * hk);
vf = (hipComplex*)malloc( sizeof(hipComplex)* min_val * n * hk);

// T take tfft
int bat = m*n;
batchedTfft(T,k,bat,tf);
hipDeviceSynchronize();

// U take tfft
bat =  m*min_val;
batchedTfft(US,k,bat,uf);
hipDeviceSynchronize();

// U*X=T
hipComplex *u, *x, *t;
hipMalloc((void**)&u, sizeof(hipComplex)* m * min_val);
hipMalloc((void**)&x, sizeof(hipComplex)* min_val * n);
hipMalloc((void**)&t, sizeof(hipComplex)* m * n);

#pragma unroll
for(int i=0;i< hk;i++){
hipMemcpy(u,uf+i*m*min_val,sizeof(hipComplex)* m * min_val ,hipMemcpyHostToDevice);
hipMemcpy(t,tf+i*m*n,sizeof(hipComplex)* m * n ,hipMemcpyHostToDevice);
qrsolve(u,t,m,min_val,n,x);
hipDeviceSynchronize();
hipMemcpy(vf+i*min_val*n,x,sizeof(hipComplex)* min_val * n,hipMemcpyDeviceToHost);
}

hipDeviceSynchronize();
if(u) hipFree(u);
if(x) hipFree(x);
if(t) hipFree(t);
// vf take Tifft
bat = min_val*n;
batchedTifft(V,k,bat,vf);
hipDeviceSynchronize();

if(tf) free(tf);
if(uf) free(uf);
if(vf) free(vf);
}
