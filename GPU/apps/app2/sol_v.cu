#include "hip/hip_runtime.h"
#include"sol_v.h"
/**
* solve tensor US*V=T
* US is a tensor , size: m * min(m,n) * k
* T is a tensor , size: m * n * k
* V is a tensor ,size: min(m,n) * n * k 
*/
void solve_v(float* T,float* US,const int m,const int n,const int k,float* V){
int hk = k/2+1;
int min = Min(m,n);
hipComplex *tf, *uf, *vf;
tf = (hipComplex*)malloc( sizeof(hipComplex)* m* n* hk);
uf = (hipComplex*)malloc( sizeof(hipComplex)* m * min * hk);
vf = (hipComplex*)malloc( sizeof(hipComplex)* min * n * hk);

// T take tfft
int bat = m*n;
batchedTfft(T,k,bat,tf);

// U take tfft
bat =  m*min;
batchedTfft(US,k,bat,uf);

// U*X=T
hipComplex *u, *x, *t;
hipMalloc((void**)&u, sizeof(hipComplex)* m * min);
hipMalloc((void**)&x, sizeof(hipComplex)* min * n);
hipMalloc((void**)&t, sizeof(hipComplex)* m * n);

#pragma unroll
for(int i=0;i< hk;i++){
hipMemcpy(u,uf+i*m*min,sizeof(hipComplex)* m * min ,hipMemcpyHostToDevice);
hipMemcpy(t,tf+i*m*n,sizeof(hipComplex)* m * n ,hipMemcpyHostToDevice);
qrsolve(u,t,m,min,n,x);
hipMemcpy(vf+i*min*n,x,sizeof(hipComplex)* min * n,hipMemcpyDeviceToHost);
}
if(u) hipFree(u);
if(x) hipFree(x);
if(t) hipFree(t);
// vf take Tifft
bat = min*n;
batchedTifft(V,k,bat,vf);

if(tf) free(tf);
if(uf) free(uf);
if(vf) free(vf);
}
